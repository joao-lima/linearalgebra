
#include <stdio.h>
#include <string.h>

#include "lb.h"
#include "cuda_safe.h"
#include "lb_kernels.cu"

#define OBJ(X,Y,N)	((Y-1)*N+(X-1))
#define POS(X,Y,N)	(Y*N+X)

static void lb_allocate( struct lattice *lb );

void lb_config( struct lattice *lb, const char *path_parameters,
		const char * path_obstacles )
{
	FILE *f_parameters, *f_obstacles;
	int c=0;
	int i, j;

#ifdef _DEBUG
	fprintf( stdout, "lb_config\n" );
	fflush(stdout);
#endif
	f_parameters= fopen( path_parameters, "r" );
	f_obstacles= fopen( path_obstacles, "r" );
	if( (f_parameters == NULL) || (f_obstacles == NULL) ) {
		fprintf( stderr, "No file found\n" );
		fflush( stderr );
		exit( EXIT_FAILURE );
	}

	fscanf( f_parameters, "%d", &lb->max_iter );
	fscanf( f_parameters, "%f", &lb->density );
	fscanf( f_parameters, "%f", &lb->accel );
	fscanf( f_parameters, "%f", &lb->omega );
	fscanf( f_parameters, "%f", &lb->r_rey );

	fscanf( f_obstacles, "%d", &lb->nx );
	fscanf( f_obstacles, "%d", &lb->ny );
	fscanf( f_obstacles, "%d", &lb->ndim );
	fscanf( f_obstacles, "%d", &lb->nobst );

	fprintf( stdout, "nx=%d ny=%d ndim=%d maxiter=%d nobst=%d\n",
	      lb->nx, lb->ny, lb->ndim, lb->max_iter, lb->nobst );
	fflush( stdout );

	lb_allocate( lb );
	while( c < lb->nobst ){
		fscanf( f_obstacles, "%d %d", &i, &j );
		lb->h_obst[ OBJ(i,j,lb->nx) ] = 1;
		c++;
	}
	fclose( f_parameters );
	fclose( f_obstacles );
}

static void lb_allocate( struct lattice *lb )
{
	unsigned int mem_size;
	int i;
	
#ifdef _DEBUG
	fprintf( stdout, "lb_allocate\n" );
	fflush(stdout);
#endif
	// memory for the lattice
	mem_size= lb->nx * lb->ny * sizeof(float);
	for( i= 0; i < lb->ndim; i++ ) {
		lb->h_f[i]= (float*) malloc( mem_size );
		CUDA_SAFE_CALL( hipMalloc( (void**)&lb->d_f[i], mem_size ) );
		CUDA_SAFE_CALL( hipMalloc( (void**)&lb->d_tf[i], mem_size ) );
	}

	// memory for obstacles
	mem_size= lb->nx * lb->ny * sizeof(unsigned short);
	lb->h_obst= (unsigned short*) malloc( mem_size );
	memset( lb->h_obst, 0, mem_size );
	CUDA_SAFE_CALL( hipMalloc( (void**)&lb->d_obst, mem_size ) );
}

void lb_init( struct lattice *lb )
{
	dim3 threads( BLOCK_SIZE, BLOCK_SIZE );
	dim3 grid( (lb->nx+BLOCK_SIZE-1)/threads.x, (lb->ny+BLOCK_SIZE-1)/threads.y );
#ifdef _DEBUG
	fprintf( stdout, "lb_init\n" );
	fflush(stdout);
#endif
	lb_init_kernel<<< grid, threads >>>( 
		lb->d_f[0], lb->d_f[1], lb->d_f[2], lb->d_f[3], lb->d_f[4], 
		lb->d_f[5], lb->d_f[6], lb->d_f[7], lb->d_f[8],
			lb->nx, lb->ny, lb->density );
	CUDA_SAFE_THREAD_SYNC();
	CUDA_SAFE_CALL( hipMemcpy( lb->d_obst, lb->h_obst,
		lb->nx * lb->ny * sizeof(unsigned short),
	       	hipMemcpyHostToDevice) );
#if 0
	int x, y, i;
	float t_0 = lb->density * 4.0 / 9.0;
	float t_1 = lb->density / 9.0;
	float t_2 = lb->density / 36.0;
	for( x= 0; x < lb->nx; x++) {
		for( y= 0; y < lb->ny; y++ ){
		//zero velocity density
		lb->h_f[0][ POS(x,y,lb->nx) ] = t_0;
		//equilibrium densities for axis speeds
		lb->h_f[1][ POS(x,y,lb->nx) ] = t_1;
		lb->h_f[2][ POS(x,y,lb->nx) ] = t_1;
		lb->h_f[3][ POS(x,y,lb->nx) ] = t_1;
		lb->h_f[4][ POS(x,y,lb->nx) ] = t_1;
		//equilibrium densities for diagonal speeds
		lb->h_f[5][ POS(x,y,lb->nx) ] = t_2;
		lb->h_f[6][ POS(x,y,lb->nx) ] = t_2;
		lb->h_f[7][ POS(x,y,lb->nx) ] = t_2;
		lb->h_f[8][ POS(x,y,lb->nx) ] = t_2;
		}
	}
	for( i= 0; i < lb->ndim; i++ )
		CUDA_SAFE_CALL( hipMemcpy( lb->d_f[i], lb->h_f[i],
			lb->nx * lb->ny * sizeof(float),
			hipMemcpyHostToDevice) );
#endif
}


float lb_velocity( struct lattice *lbm, int time )
{
	(void) lbm;
	(void) time;
#if 0
	int x, y, n_free;
	float u_x, d_loc;

	x = nx/2;
	n_free = 0;
	u_x = 0;

	for( y = 0; y < ny; y++ ) {
		if ( obst[pos(x,y)] == false ){
			d_loc = f0[pos(x,y)];
			d_loc += f1[pos(x,y)];
			d_loc += f2[pos(x,y)];
			d_loc += f3[pos(x,y)];
			d_loc += f4[pos(x,y)];
			d_loc += f5[pos(x,y)];
			d_loc += f6[pos(x,y)];
			d_loc += f7[pos(x,y)];
			d_loc += f8[pos(x,y)];
			u_x = u_x + (f1[pos(x,y)]
				 + f5[pos(x,y)] + f8[pos(x,y)] - 
				 (f3[pos(x,y)] + f6[pos(x,y)]
				  + f7[pos(x,y)])) / d_loc;
			n_free++;
		}
	}
	/*
	//Optional
	if (time%500 == 0) {
		FILE *c = fopen("convergence9.out", "a");
		fprintf(c, "%d %lf\n", time, u_x / n_free);
		fclose(c);
	}
	*/
	return u_x / n_free;
#endif
	return 0;
}

void lb_redistribute( struct lattice *lb )
{
	dim3 threads( BLOCK_SIZE, 1 );
	dim3 grid( (lb->ny+BLOCK_SIZE-1)/BLOCK_SIZE, 1 );
#ifdef _DEBUG
	fprintf( stdout, "lb_redistribute\n" );
	fflush(stdout);
#endif
	lb_redistribute_kernel<<< grid, threads >>>(
		lb->d_f[0], lb->d_f[1], lb->d_f[2], lb->d_f[3], lb->d_f[4], 
		lb->d_f[5], lb->d_f[6], lb->d_f[7], lb->d_f[8], lb->d_obst,
		lb->accel, lb->density, lb->nx, lb->ny );
	CUDA_SAFE_THREAD_SYNC();
}

void lb_propagate( struct lattice *lb )
{
	dim3 threads( BLOCK_SIZE, BLOCK_SIZE );
	dim3 grid( (lb->nx+BLOCK_SIZE-1)/threads.x,
			(lb->ny+BLOCK_SIZE-1)/threads.y );
#ifdef _DEBUG
	fprintf( stdout, "lb_propagate\n" );
	fflush(stdout);
#endif
	lb_propagate_kernel<<< grid, threads >>>( 
		lb->d_f[0], lb->d_f[1], lb->d_f[2], lb->d_f[3], lb->d_f[4], 
		lb->d_f[5], lb->d_f[6], lb->d_f[7], lb->d_f[8], 
		lb->d_tf[0], lb->d_tf[1], lb->d_tf[2], lb->d_tf[3],
		lb->d_tf[4], lb->d_tf[5], lb->d_tf[6], lb->d_tf[7],
		lb->d_tf[8], lb->nx, lb->ny );
	CUDA_SAFE_THREAD_SYNC();
}

void lb_bounceback( struct lattice *lb )
{
	dim3 threads( BLOCK_SIZE, BLOCK_SIZE );
	dim3 grid( (lb->nx+BLOCK_SIZE-1)/threads.x, (lb->ny+BLOCK_SIZE-1)/threads.y );
#ifdef _DEBUG
	fprintf( stdout, "lb_bounceback\n" );
	fflush(stdout);
#endif
	lb_bounceback_kernel<<< grid, threads >>>( lb->d_f[0], lb->d_f[1], 
		lb->d_f[2], lb->d_f[3], lb->d_f[4], lb->d_f[5], lb->d_f[6], 
		lb->d_f[7], lb->d_f[8],
		lb->d_tf[0], lb->d_tf[1], lb->d_tf[2], lb->d_tf[3], 
		lb->d_tf[4], lb->d_tf[5], lb->d_tf[6], lb->d_tf[7],
		lb->d_tf[8], lb->d_obst, lb->nx, lb->ny );
	CUDA_SAFE_THREAD_SYNC();
}

void lb_relaxation( struct lattice *lb )
{
#if 1
	/* here a kernel call */
	dim3 threads( BLOCK_SIZE, BLOCK_SIZE );
	dim3 grid( (lb->nx+BLOCK_SIZE-1)/threads.x,
			(lb->ny+BLOCK_SIZE-1)/threads.y );
	lb_relaxation_kernel<<< grid, threads >>>( lb->d_f[0], lb->d_f[1], 
		lb->d_f[2], lb->d_f[3], lb->d_f[4], lb->d_f[5], lb->d_f[6], 
		lb->d_f[7], lb->d_f[8],
		lb->d_tf[0], lb->d_tf[1], lb->d_tf[2], lb->d_tf[3], 
		lb->d_tf[4], lb->d_tf[5], lb->d_tf[6], lb->d_tf[7], 
		lb->d_tf[8], lb->d_obst,
		lb->nx, lb->ny, lb->omega );
	CUDA_SAFE_THREAD_SYNC();
#endif
}

void lb_finalize( struct lattice *lb )
{
	int i;
#ifdef _DEBUG
	fprintf( stdout, "lb_finalize\n" );
	fflush(stdout);
#endif
	for( i= 0; i < lb->ndim; i++ )
		CUDA_SAFE_CALL( hipMemcpy( lb->h_f[i], lb->d_f[i],
			lb->nx * lb->ny * sizeof(float),
			hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL( hipDeviceSynchronize() );
}

void lb_write_results( struct lattice *lb, const char *output )
{
	int x, y, i;
	int obsval;
	float u_x, u_y, d_loc, press;
	//Square speed of sound
	float c_squ = 1.0 / 3.0;

#ifdef _DEBUG
	fprintf( stdout, "lb_write_results\n" );
	fflush(stdout);
#endif
	//Open results output file
	FILE *archive = fopen( output , "w");

	//write results
	fprintf( archive, "VARIABLES = X, Y, VX, VY, PRESS, OBST\n" );
	fprintf( archive,"ZONE I= %d, J= %d, F=POINT\n", lb->nx, lb->ny );

	for( y = 0; y < lb->ny; y++ ){
		for( x = 0; x < lb->nx; x++ ){
			//if obstacle node, nothing is to do
			if ( lb->h_obst[POS(x,y,lb->nx)] == 1 ) {
				//obstacle indicator
				obsval = 1;
				//velocity components = 0
				u_x = 0.0;
				u_y = 0.0;
				//pressure = average pressure
				press = lb->density * c_squ;
			} else {
				//integral local density
				//initialize variable d_loc
				d_loc= 0.0f;
				for( i= 0; i < lb->ndim; i++ )
					d_loc += lb->h_f[i][ POS(x,y,lb->nx) ];

#define NODE(X,Y,D)		(lb->h_f[D][(Y*lb->nx+X)])
				// x-, and y- velocity components
				u_x = (NODE(x,y,1) + NODE(x,y,5) + NODE(x,y,8) - (NODE(x,y,3) + NODE(x,y,6) + NODE(x,y,7))) / d_loc;
				u_y = (NODE(x,y,2) + NODE(x,y,5) + NODE(x,y,6) - (NODE(x,y,4) + NODE(x,y,7) + NODE(x,y,8))) / d_loc;
				
				//pressure
				press = d_loc * c_squ;
				obsval = 0;
			}
			fprintf( archive, "%d %d %f %f %f %d\n", x, y, u_x,
				       	u_y, press, obsval );
		}
	}
	
	fclose(archive);
}

void lb_free( struct lattice *lb )
{
#ifdef _DEBUG
	fprintf( stdout, "lb_free\n" );
	fflush(stdout);
#endif
	int i;
	for( i= 0; i < lb->ndim; i++ ) {
		CUDA_SAFE_CALL( hipFree( lb->d_f[i] ) );
		CUDA_SAFE_CALL( hipFree( lb->d_tf[i] ) );
		free( lb->h_f[i] );
	}

	free( lb->h_obst );
	CUDA_SAFE_CALL( hipFree( lb->d_obst ) );
	CUDA_SAFE_CALL( hipDeviceReset() );
#ifdef _DEBUG
	fprintf(stdout,"bazzinga!\n"); fflush(stdout);
#endif
}
