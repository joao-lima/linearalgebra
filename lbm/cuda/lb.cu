#include "hip/hip_runtime.h"

#include <stdio.h>
#include <memset.h>

#include "lb.h"

static void lb_allocate( struct lattice *lb );

__device__ __host__ inline
unsigned int pos( const int x, const int y,
		const int n ) 
{
	return ( y * n + x );
}

void lb_config( struct lattice *lb, const char *path_parameters,
		const char * path_obstacles )
{
	FILE *f_parameters, *f_obstacles;
	int max, c=0;
	int i, j;

	f_parameters= fopen( path_parameters, "r" );
	f_obstacles= fopen( path_obstacles, "r" );
	if( (f_parameters == NULL) || (f_obstacles == NULL) )
		return;

	fscanf( f_parameters, "%d", &lb->max_iter );
	fscanf( f_parameters, "%f", &lb->density );
	fscanf( f_parameters, "%f", &lb->accel );
	fscanf( f_parameters, "%f", &lb->omega );
	fscanf( f_parameters, "%f", &lb->r_rey );

	fscanf( f_obstacles, "%d", &lb->nx );
	fscanf( f_obstacles, "%d", &lb->ny );
	fscanf( f_obstacles, "%d", &lb->ndim );
	fscanf( f_obstacles, "%d", &lb->nobst );

	lb_allocate( &lb );
	while( c < lb->nobst ){
		fscanf( f_obstacles, "%d %d", &i, &j );
		// TODO: problema aqui com indices e entrada
		lb->h_obst[ pos(i,j) ] = 1;
		c++;
	}
	fclose( f_parameters );
	fclose( f_obstacles );
}

static void lb_allocate( struct lattice *lb )
{
	unsigned int memsize;
	
	// memory for the lattice
	memsize= lb->nx * lb->ny * sizeof(struct lb_d2q9);
	CUDA_SAFE_CALL( hipMalloc( (void**)&lb->d_data, memsize ) );
	CUDA_SAFE_CALL( hipMemset( (void**)&lb->d_data, 0, memsize ) );
	CUDA_SAFE_CALL( hipMalloc( (void**)&lb->d_tmp, memsize ) );
	CUDA_SAFE_CALL( hipMemset( (void**)&lb->d_tmp, 0, memsize ) );

	// memory for obstacles
	memsize= lb->nobst * sizeof(int);
	lb->h_obst= (int*)malloc( memsize );
	memset( lb->h_obst, 0, memsize );
	CUDA_SAFE_CALL( hipMalloc( (void**)&lb->d_obst, memsize ) );
}

void lb_init( struct lattice *lb )
{
	dim3 threads( BLOCK_SIZE, BLOCK_SIZE );
	dim3 grid( (lb->nx+BLOCK_SIZE-1)/threads.x,
			(lb->ny+BLOCK_SIZE-1)/threads.y );
	lb_init_kernel<<< grid, threads >>>( lb->d_data, lb->nx, lb->ny );
}


/* essa função pode ter uma implementação CUDA/thrust 
   eu vi uma função chamada transform_reduce, quem sabe ...
*/
float lb_velocity( struct lattice *lbm, int time )
{
#if 0
	int x, y, n_free;
	float u_x, d_loc;

	x = nx/2;
	n_free = 0;
	u_x = 0;

	for( y = 0; y < ny; y++ ) {
		if ( obst[pos(x,y)] == false ){
			d_loc = f0[pos(x,y)];
			d_loc += f1[pos(x,y)];
			d_loc += f2[pos(x,y)];
			d_loc += f3[pos(x,y)];
			d_loc += f4[pos(x,y)];
			d_loc += f5[pos(x,y)];
			d_loc += f6[pos(x,y)];
			d_loc += f7[pos(x,y)];
			d_loc += f8[pos(x,y)];
			u_x = u_x + (f1[pos(x,y)]
				 + f5[pos(x,y)] + f8[pos(x,y)] - 
				 (f3[pos(x,y)] + f6[pos(x,y)]
				  + f7[pos(x,y)])) / d_loc;
			n_free++;
		}
	}
	/*
	//Optional
	if (time%500 == 0) {
		FILE *c = fopen("convergence9.out", "a");
		fprintf(c, "%d %lf\n", time, u_x / n_free);
		fclose(c);
	}
	*/
	return u_x / n_free;
#endif
	return 0;
}

void lb_redistribute( struct lattice *lb );
{
	/* here a kernel call */
	// tem de chamar esse kernel com uma dimensao apenas
	dim3 threads( BLOCK_SIZE, 1 );
	dim3 grid( (lb->ny+BLOCK_SIZE-1)/BLOCK_SIZE, 1 );
	redistribute_kernel<<< grid, threads >>>( lb->d_data, lb->d_obst,
		lb->accel, lb->density, lb->nx, lb->ny );
}

/*
	PROPAGATE kernel
	Authors: Joao
*/

void lb_propagate( struct lattice *lb )
{
	/* here a kernel call */
	dim3 threads( BLOCK_SIZE, BLOCK_SIZE );
	dim3 grid( (lb->nx+BLOCK_SIZE-1)/threads.x,
			(lb->ny+BLOCK_SIZE-1)/threads.y );
	lb_propagate_kernel<<< grid, threads >>>( lb->d_data, lb->d_tmp,
		lb->nx, lb->ny );
}

void lb_bounceback( struct lattice *lb )
{
	/* here a kernel call */
	dim3 threads( BLOCK_SIZE, BLOCK_SIZE );
	dim3 grid( (nx+BLOCK_SIZE-1)/threads.x, (ny+BLOCK_SIZE-1)/threads.y );
	lb_bounceback_kernel<<< grid, threads >>>( lb->d_data, lb->d_tmp,
			lb->d_obst, lb->nx, lb->ny );
}

/*
	RELAXATION kernel
	Authors: Antonio, Catia e Joao
*/
__global__ void relaxation_kernel( 
	float *f0, float *f1, float *f2, float *f3, float *f4, float *f5,
	float *f6, float *f7, float *f8,
	float *tf0, float *tf1, float *tf2, float *tf3, float *tf4, 
	float *tf5, float *tf6, float *tf7, float *tf8, bool* obst,
	int nx, int ny, float omega )
{
	//local variables
	float c_squ = 1.0 / 3.0;
	float t_0 = 4.0 / 9.0;
	float t_1 = 1.0 / 9.0;
	float t_2 = 1.0 / 36.0;
	float u_x, u_y;
	float u_n[9], n_equ[9], u_squ, d_loc;
	int y = blockIdx.y * blockDim.y + threadIdx.y; 
	int x = blockIdx.x * blockDim.x + threadIdx.x; 

	if( (y >= ny) || (x >= nx) ) return;
	if ( obst[pos(x,y,nx)] == false ) {
		d_loc = tf0[pos(x,y,nx)];
		d_loc += tf1[pos(x,y,nx)];
		d_loc += tf2[pos(x,y,nx)];
		d_loc += tf3[pos(x,y,nx)];
		d_loc += tf4[pos(x,y,nx)];
		d_loc += tf5[pos(x,y,nx)];
		d_loc += tf6[pos(x,y,nx)];
		d_loc += tf7[pos(x,y,nx)];
		d_loc += tf8[pos(x,y,nx)];

		//x-, and y- velocity components
		u_x = (tf1[pos(x,y,nx)] + tf5[pos(x,y,nx)] + tf8[pos(x,y,nx)] -
				(tf3[pos(x,y,nx)] + tf6[pos(x,y,nx)] +
				 tf7[pos(x,y,nx)])) / d_loc;
		//u_x = (l->temp[x][y][1] + l->temp[x][y][5] + l->temp[x][y][8] - (l->temp[x][y][3] + l->temp[x][y][6] + l->temp[x][y][7])) / d_loc;

		u_y = (tf2[pos(x,y,nx)] + tf5[pos(x,y,nx)] + tf6[pos(x,y,nx)] -
				(tf4[pos(x,y,nx)] + tf7[pos(x,y,nx)] +
				 tf8[pos(x,y,nx)])) / d_loc;
		//u_y = (l->temp[x][y][2] + l->temp[x][y][5] + l->temp[x][y][6] - (l->temp[x][y][4] + l->temp[x][y][7] + l->temp[x][y][8])) / d_loc;

		//square velocity
		u_squ = u_x * u_x + u_y * u_y;

		//n- velocity compnents
		//only 3 speeds would be necessary
		u_n[1] = u_x;
		u_n[2] = u_y;
		u_n[3] = -u_x;
		u_n[4] = -u_y;
		u_n[5] = u_x + u_y;
		u_n[6] = -u_x + u_y;
		u_n[7] = -u_x - u_y;
		u_n[8] = u_x - u_y;
		
		//zero velocity density
		n_equ[0] = t_0 * d_loc * (1.0 - u_squ / (2.0 * c_squ));
		//axis speeds: factor: t_1
		n_equ[1] = t_1 * d_loc * (1.0 + u_n[1] / c_squ + u_n[1] * u_n[1] / (2.0 * c_squ * c_squ) - u_squ / (2.0 * c_squ));
		n_equ[2] = t_1 * d_loc * (1.0 + u_n[2] / c_squ + u_n[2] * u_n[2] / (2.0 * c_squ * c_squ) - u_squ / (2.0 * c_squ));
		n_equ[3] = t_1 * d_loc * (1.0 + u_n[3] / c_squ + u_n[3] * u_n[3] / (2.0 * c_squ * c_squ) - u_squ / (2.0 * c_squ));
		n_equ[4] = t_1 * d_loc * (1.0 + u_n[4] / c_squ + u_n[4] * u_n[4] / (2.0 * c_squ * c_squ) - u_squ / (2.0 * c_squ));

		//diagonal speeds: factor t_2
		n_equ[5] = t_2 * d_loc * (1.0 + u_n[5] / c_squ + u_n[5] * u_n[5] / (2.0 * c_squ * c_squ) - u_squ / (2.0 * c_squ));
		n_equ[6] = t_2 * d_loc * (1.0 + u_n[6] / c_squ + u_n[6] * u_n[6] / (2.0 * c_squ * c_squ) - u_squ / (2.0 * c_squ));
		n_equ[7] = t_2 * d_loc * (1.0 + u_n[7] / c_squ + u_n[7] * u_n[7] / (2.0 * c_squ * c_squ) - u_squ / (2.0 * c_squ));
		n_equ[8] = t_2 * d_loc * (1.0 + u_n[8] / c_squ + u_n[8] * u_n[8] / (2.0 * c_squ * c_squ) - u_squ / (2.0 * c_squ));

		
		//relaxation step
		f0[pos(x,y,nx)] = tf0[pos(x,y,nx)] + omega * (n_equ[0] - tf0[pos(x,y,nx)]);
		f1[pos(x,y,nx)] = tf1[pos(x,y,nx)] + omega * (n_equ[1] - tf1[pos(x,y,nx)]);
		f2[pos(x,y,nx)] = tf2[pos(x,y,nx)] + omega * (n_equ[2] - tf2[pos(x,y,nx)]);
		f3[pos(x,y,nx)] = tf3[pos(x,y,nx)] + omega * (n_equ[3] - tf3[pos(x,y,nx)]);
		f4[pos(x,y,nx)] = tf4[pos(x,y,nx)] + omega * (n_equ[4] - tf4[pos(x,y,nx)]);
		f5[pos(x,y,nx)] = tf5[pos(x,y,nx)] + omega * (n_equ[5] - tf5[pos(x,y,nx)]);
		f6[pos(x,y,nx)] = tf6[pos(x,y,nx)] + omega * (n_equ[6] - tf6[pos(x,y,nx)]);
		f7[pos(x,y,nx)] = tf7[pos(x,y,nx)] + omega * (n_equ[7] - tf7[pos(x,y,nx)]);
		f8[pos(x,y,nx)] = tf8[pos(x,y,nx)] + omega * (n_equ[8] - tf8[pos(x,y,nx)]);
		//for (i = 0; i < l->n; i++) {
		//	l->node[x][y][i] = l->temp[x][y][i] + omega * (n_equ[i] - l->temp[x][y][i]);
		//}	
	}
}

void lb::relaxation( void )
{
	/* here a kernel call */
	dim3 threads( BLOCK_SIZE, BLOCK_SIZE );
	dim3 grid( (nx+BLOCK_SIZE-1)/threads.x, (ny+BLOCK_SIZE-1)/threads.y );
	relaxation_kernel<<< grid, threads >>>(
		thrust::raw_pointer_cast(&d_f0[0]),
		thrust::raw_pointer_cast(&d_f1[0]),
		thrust::raw_pointer_cast(&d_f2[0]),
		thrust::raw_pointer_cast(&d_f3[0]),
		thrust::raw_pointer_cast(&d_f4[0]),
		thrust::raw_pointer_cast(&d_f5[0]),
		thrust::raw_pointer_cast(&d_f6[0]),
		thrust::raw_pointer_cast(&d_f7[0]),
		thrust::raw_pointer_cast(&d_f8[0]),
		// temps from here
		thrust::raw_pointer_cast(&d_tf0[0]),
		thrust::raw_pointer_cast(&d_tf1[0]),
		thrust::raw_pointer_cast(&d_tf2[0]),
		thrust::raw_pointer_cast(&d_tf3[0]),
		thrust::raw_pointer_cast(&d_tf4[0]),
		thrust::raw_pointer_cast(&d_tf5[0]),
		thrust::raw_pointer_cast(&d_tf6[0]),
		thrust::raw_pointer_cast(&d_tf7[0]),
		thrust::raw_pointer_cast(&d_tf8[0]),
		// others
		thrust::raw_pointer_cast(&d_obst[0]),
		nx, ny, omega );
}

void lb::write_results( const char *file ) 
{
	//local variables
	int x, y;
	bool obsval;
	float u_x, u_y, d_loc, press;

	//Square speed of sound
	float c_squ = 1.0 / 3.0;

	//Open results output file
	FILE *archive = fopen(file, "w");

	//write results
	fprintf( archive, "VARIABLES = X, Y, VX, VY, PRESS, OBST\n" );
	fprintf( archive,"ZONE I= %d, J= %d, F=POINT\n", nx, ny );

	thrust::copy( d_f0.begin(), d_f0.end(), f0.begin() );
	thrust::copy( d_f1.begin(), d_f1.end(), f1.begin() );
	thrust::copy( d_f2.begin(), d_f2.end(), f2.begin() );
	thrust::copy( d_f3.begin(), d_f3.end(), f3.begin() );
	thrust::copy( d_f4.begin(), d_f4.end(), f4.begin() );
	thrust::copy( d_f5.begin(), d_f5.end(), f5.begin() );
	thrust::copy( d_f6.begin(), d_f6.end(), f6.begin() );
	thrust::copy( d_f7.begin(), d_f7.end(), f7.begin() );
	thrust::copy( d_f8.begin(), d_f8.end(), f8.begin() );
	for( y = 0; y < ny; y++ ){
		for( x = 0; x < nx; x++ ){
			//if obstacle node, nothing is to do
			if (obst[pos(x,y)] == true) {
				//obstacle indicator
				obsval = true;
				//velocity components = 0
				u_x = 0.0;
				u_y = 0.0;
				//pressure = average pressure
				press = density * c_squ;
			} else {
				//integral local density
				//initialize variable d_loc
				//d_loc = 0.0;
				//for (i = 0; i < 9; i++) {
				//	d_loc += l->node[x][y][i];
				//}
				d_loc = f0[pos(x,y)];
				d_loc += f1[pos(x,y)];
				d_loc += f2[pos(x,y)];
				d_loc += f3[pos(x,y)];
				d_loc += f4[pos(x,y)];
				d_loc += f5[pos(x,y)];
				d_loc += f6[pos(x,y)];
				d_loc += f7[pos(x,y)];
				d_loc += f8[pos(x,y)];
				// x-, and y- velocity components
				u_x = (f1[pos(x,y)] + f5[pos(x,y)] + f8[pos(x,y)] - (f3[pos(x,y)] + f6[pos(x,y)] + f7[pos(x,y)])) / d_loc;
				//u_x = (l->node[x][y][1] + l->node[x][y][5] + l->node[x][y][8] - (l->node[x][y][3] + l->node[x][y][6] + l->node[x][y][7])) / d_loc;
				u_y = (f2[pos(x,y)] + f5[pos(x,y)] + f6[pos(x,y)] - (f4[pos(x,y)] + f7[pos(x,y)] + f8[pos(x,y)])) / d_loc;

				//u_y = (l->node[x][y][2] + l->node[x][y][5] + l->node[x][y][6] - (l->node[x][y][4] + l->node[x][y][7] + l->node[x][y][8])) / d_loc;
				
				//pressure
				press = d_loc * c_squ;
				obsval = false;
			}
			fprintf( archive, "%d %d %f %f %f %d\n", x, y, u_x,
				       	u_y, press, obsval );
		}
	}
	
	fclose(archive);
}

