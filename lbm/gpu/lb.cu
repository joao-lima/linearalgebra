#include "hip/hip_runtime.h"

#include <iostream>
#include <fstream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>

#include "lb.h"

lb::lb() {}

void lb::read( const char *parameters, const char *obstacles )
{
	std::ifstream par, obs;
	int max, c=0;
	int i, j;

	par.open( parameters );
	obs.open( obstacles );
	if( !par.is_open() || !obs.is_open() )
		return;
	par >> max_iter;
	par >> density;
	par >> accel;
	par >> omega;
	par >> r_rey;

	obs >> nx;
	obs >> ny;
	obs >> ndim;
	obs >> max;

	std::cout << "nx=" << nx << " ny=" << ny << " ndim=" << ndim 
		<< std::endl;
	resize( nx * ny );
	while( c < max ){
		obs >> i;
		obs >> j;
		obst[pos(i-1,j-1)] = true;
		c++;
	}
	par.close();
	obs.close();
}

void lb::resize( const int n )
{
	f0.resize( n ); f1.resize( n ); f2.resize( n ); f3.resize( n );
       	f4.resize( n ); f5.resize( n ); f6.resize( n ); f7.resize( n );
       	f8.resize( n );
	/*
	d_f0.resize( n ); d_f1.resize( n ); d_f2.resize( n ); d_f3.resize( n );
       	d_f4.resize( n ); d_f5.resize( n ); d_f6.resize( n ); d_f7.resize( n );
       	d_f8.resize( n );
	*/
	d_tf0.resize( n ); d_tf1.resize( n ); d_tf2.resize( n ); d_tf3.resize( n );
       	d_tf4.resize( n ); d_tf5.resize( n ); d_tf6.resize( n ); d_tf7.resize( n );
       	d_tf8.resize( n );
	obst.resize( n );
	//d_obst.resize( n );
}

void lb::init( )
{
	int x, y;
	float t_0 = density * 4.0 / 9.0;
	float t_1 = density / 9.0;
	float t_2 = density / 36.0;

	//loop over computational domain
	for (x = 0; x < nx; x++) {
		for (y = 0; y < ny; y++) {
			//zero velocity density
			f0[pos(x,y)] = t_0;
			//equilibrium densities for axis speeds
			f1[pos(x,y)] = t_1;
			f2[pos(x,y)] = t_1;
			f3[pos(x,y)] = t_1;
			f4[pos(x,y)] = t_1;
			//equilibrium densities for diagonal speeds
			f5[pos(x,y)] = t_2;
			f6[pos(x,y)] = t_2;
			f7[pos(x,y)] = t_2;
			f8[pos(x,y)] = t_2;
		}
	}
	// Copy of host to device
	d_f0= f0; d_f1= f1; d_f2= f2; d_f3= f3; d_f4= f4; d_f5= f5; d_f6= f6;
	d_f7= f7; d_f8= f8;
	d_obst= obst;
}

/* essa função pode ter uma implementação CUDA/thrust 
   eu vi uma função chamada transform_reduce, quem sabe ...
*/
float lb::velocity( int time ) 
{
	int x, y, n_free;
	float u_x, d_loc;

	x = nx/2;
	n_free = 0;
	u_x = 0;

	for( y = 0; y < ny; y++ ) {
		if ( obst[pos(x,y)] == false ){
			d_loc = f0[pos(x,y)];
			d_loc += d_loc + f1[pos(x,y)];
			d_loc += d_loc + f2[pos(x,y)];
			d_loc += d_loc + f3[pos(x,y)];
			d_loc += d_loc + f4[pos(x,y)];
			d_loc += d_loc + f5[pos(x,y)];
			d_loc += d_loc + f6[pos(x,y)];
			d_loc += d_loc + f7[pos(x,y)];
			d_loc += d_loc + f8[pos(x,y)];
			u_x = u_x + (f1[pos(x,y)]
				 + f5[pos(x,y)] + f8[pos(x,y)] - 
				 (f3[pos(x,y)] + f6[pos(x,y)]
				  + f7[pos(x,y)])) / d_loc;
			n_free++;
		}
	}
	/*
	//Optional
	if (time%500 == 0) {
		FILE *c = fopen("convergence9.out", "a");
		fprintf(c, "%d %lf\n", time, u_x / n_free);
		fclose(c);
	}
	*/
	return u_x / n_free;
}

__global__ void redistribute_kernel( float * f1, float * f3, float * f5, 
	float * f6,float * f7,float * f8, bool* obst, float accel,
       	float density, int nx, int ny ) {
    //nx e ny sao as dimensoes
    //local variables
    float t_1 = density * accel / 9.0;
    float t_2 = density * accel / 36.0;

    int row = blockIdx.x * blockDim.x + threadIdx.x; 
    if (row > ny) return;
    //for (y = 0; y < l->ly; y++) {
    //check to avoid negative densities
    //check false | true
    if ( (obst[row * nx] == false) && ((f3[row * nx] - t_1) > 0) && 
                 ((f6[row * nx] - t_2) > 0) && (f7[row * nx] - t_2 > 0)) {
      //increase east
      f1[row * nx] += t_1;
      //l->node[0][y][1] += t_1;
      //decrease west
      f3[row * nx] -= t_1;
      //l->node[0][y][3] -= t_1;
      //increase north-east
      f5[row * nx] += t_2;
      //l->node[0][y][5] += t_2;
      //decrease north-west
      f6[row * nx] -= t_2;
      //l->node[0][y][6] -= t_2;
      //decrease south-west
      f7[row * nx] -= t_2;
      //l->node[0][y][7] -= t_2;
      //increase south-east
      f8[row * nx] += t_2;
      //l->node[0][y][8] += t_2;
    }
  //}
}

void lb::redistribute( void )
{
	/* here a kernel call */
	// tem de chamar esse kernel com uma dimensao apenas
	dim3 threads( BLOCK_SIZE, 1 );
	dim3 grid( (ny+BLOCK_SIZE-1)/BLOCK_SIZE, 1 );
	redistribute_kernel<<< grid, threads >>>(
		thrust::raw_pointer_cast(&d_f1[0]),
		thrust::raw_pointer_cast(&d_f3[0]),
		thrust::raw_pointer_cast(&d_f5[0]),
		thrust::raw_pointer_cast(&d_f6[0]),
		thrust::raw_pointer_cast(&d_f7[0]),
		thrust::raw_pointer_cast(&d_f8[0]),
		thrust::raw_pointer_cast(&d_obst[0]),
		accel, density, nx, ny );
}

void lb::propagate( void )
{
	/* here a kernel call */
}

__global__ void bounceback_kernel( float * f1, float * f2, float * f3,
		float * f4, float * f5, float * f6, float * f7, float * f8,
		float * tf1, float * tf2, float * tf3, float * tf4, 
		float * tf5, float * tf6, float * tf7, float * tf8,
		bool* obst, int nx, int ny) {
  //local variables
  //TODO ver o acesso a memoria. nao fica totalmente desalinhado usando 8 vetores nao?
  //-- indexes
  int row = blockIdx.y * blockDim.y + threadIdx.y; 
  int col = blockIdx.x * blockDim.x + threadIdx.x; 

      if ( (row > ny) || (col > nx) ) return;

      if ( obst[row * nx + col] ){
        //east
        f1[row * nx + col] = tf3[row * nx + col];
        //north
        f2[row * nx + col] = tf4[row * nx + col];
        //west
        f3[row * nx + col] = tf1[row * nx + col];
        //south
        f4[row * nx + col] = tf2[row * nx + col];
        //north-east
        f5[row * nx + col] = tf7[row * nx + col];
        //north-west
        f6[row * nx + col] = tf8[row * nx + col];
        //south-west
        f7[row * nx + col] = tf5[row * nx + col];
        //south-east
        f8[row * nx + col] = tf6[row * nx + col];
      }
}

void lb::bounceback( void )
{
	/* here a kernel call */
	dim3 threads( BLOCK_SIZE, BLOCK_SIZE );
	dim3 grid( (nx+BLOCK_SIZE-1)/threads.x, (ny+BLOCK_SIZE-1)/threads.y );
	bounceback_kernel<<< grid, threads >>>(
		thrust::raw_pointer_cast(&d_f1[0]),
		thrust::raw_pointer_cast(&d_f2[0]),
		thrust::raw_pointer_cast(&d_f3[0]),
		thrust::raw_pointer_cast(&d_f4[0]),
		thrust::raw_pointer_cast(&d_f5[0]),
		thrust::raw_pointer_cast(&d_f6[0]),
		thrust::raw_pointer_cast(&d_f7[0]),
		thrust::raw_pointer_cast(&d_f8[0]),
		// temps from here
		thrust::raw_pointer_cast(&d_tf1[0]),
		thrust::raw_pointer_cast(&d_tf2[0]),
		thrust::raw_pointer_cast(&d_tf3[0]),
		thrust::raw_pointer_cast(&d_tf4[0]),
		thrust::raw_pointer_cast(&d_tf5[0]),
		thrust::raw_pointer_cast(&d_tf6[0]),
		thrust::raw_pointer_cast(&d_tf7[0]),
		thrust::raw_pointer_cast(&d_tf8[0]),
		// others
		thrust::raw_pointer_cast(&d_obst[0]),
		nx, ny );
}

__device__ inline unsigned int pos( const int x, const int y,
		const int n ) const
{
	return ( x*n + y );
}

__global__ void relaxation_kernel( 
	float *f0, float *f1, float *f2, float *f3, float *f4, float *f5,
	float *f6, float *f7, float *f8,
	float *tf0, float *tf1, float *tf2, float *tf3, float *tf4, 
	float *tf5, float *tf6, float *tf7, float *tf8, bool* obst,
	int nx, int ny, float omega )
{
	//local variables
	float c_squ = 1.0 / 3.0;
	float t_0 = 4.0 / 9.0;
	float t_1 = 1.0 / 9.0;
	float t_2 = 1.0 / 36.0;
	float u_x, u_y;
	float u_n[9], n_equ[9], u_squ, d_loc;
	int y = blockIdx.y * blockDim.y + threadIdx.y; 
	int x = blockIdx.x * blockDim.x + threadIdx.x; 

	if( (y > ny) || (x > nx) ) return;
	if ( obst[pos(x,y,ny)] == false ) {
		d_loc = tf0[pos(x,y,ny)];
		d_loc += d_loc + tf1[pos(x,y,ny)];
		d_loc += d_loc + tf2[pos(x,y,ny)];
		d_loc += d_loc + tf3[pos(x,y,ny)];
		d_loc += d_loc + tf4[pos(x,y,ny)];
		d_loc += d_loc + tf5[pos(x,y,ny)];
		d_loc += d_loc + tf6[pos(x,y,ny)];
		d_loc += d_loc + tf7[pos(x,y,ny)];
		d_loc += d_loc + tf8[pos(x,y,ny)];

		//x-, and y- velocity components
		u_x = (tf1[pos(x,y,ny)] + tf5[pos(x,y,ny)] + tf8[pos(x,y,ny)] - (tf3[pos(x,y,ny)] + tf6[pos(x,y,ny)] + tf7[pos(x,y,ny)])) / d_loc;
		//u_x = (l->temp[x][y][1] + l->temp[x][y][5] + l->temp[x][y][8] - (l->temp[x][y][3] + l->temp[x][y][6] + l->temp[x][y][7])) / d_loc;

		u_y = (tf2[pos(x,y,ny)] + tf5[pos(x,y,ny)] + tf6[pos(x,y,ny)] - (tf4[pos(x,y,ny)] + tf7[pos(x,y,ny)] + tf8[pos(x,y,ny)])) / d_loc;
		//u_y = (l->temp[x][y][2] + l->temp[x][y][5] + l->temp[x][y][6] - (l->temp[x][y][4] + l->temp[x][y][7] + l->temp[x][y][8])) / d_loc;

		//square velocity
		u_squ = u_x * u_x + u_y * u_y;

		//n- velocity compnents
		//only 3 speeds would be necessary
		u_n[1] = u_x;
		u_n[2] = u_y;
		u_n[3] = -u_x;
		u_n[4] = -u_y;
		u_n[5] = u_x + u_y;
		u_n[6] = -u_x + u_y;
		u_n[7] = -u_x - u_y;
		u_n[8] = u_x - u_y;
		
		//zero velocity density
		n_equ[0] = t_0 * d_loc * (1.0 - u_squ / (2.0 * c_squ));
		//axis speeds: factor: t_1
		n_equ[1] = t_1 * d_loc * (1.0 + u_n[1] / c_squ + u_n[1] * u_n[1] / (2.0 * c_squ * c_squ) - u_squ / (2.0 * c_squ));
		n_equ[2] = t_1 * d_loc * (1.0 + u_n[2] / c_squ + u_n[2] * u_n[2] / (2.0 * c_squ * c_squ) - u_squ / (2.0 * c_squ));
		n_equ[3] = t_1 * d_loc * (1.0 + u_n[3] / c_squ + u_n[3] * u_n[3] / (2.0 * c_squ * c_squ) - u_squ / (2.0 * c_squ));
		n_equ[4] = t_1 * d_loc * (1.0 + u_n[4] / c_squ + u_n[4] * u_n[4] / (2.0 * c_squ * c_squ) - u_squ / (2.0 * c_squ));

		//diagonal speeds: factor t_2
		n_equ[5] = t_2 * d_loc * (1.0 + u_n[5] / c_squ + u_n[5] * u_n[5] / (2.0 * c_squ * c_squ) - u_squ / (2.0 * c_squ));
		n_equ[6] = t_2 * d_loc * (1.0 + u_n[6] / c_squ + u_n[6] * u_n[6] / (2.0 * c_squ * c_squ) - u_squ / (2.0 * c_squ));
		n_equ[7] = t_2 * d_loc * (1.0 + u_n[7] / c_squ + u_n[7] * u_n[7] / (2.0 * c_squ * c_squ) - u_squ / (2.0 * c_squ));
		n_equ[8] = t_2 * d_loc * (1.0 + u_n[8] / c_squ + u_n[8] * u_n[8] / (2.0 * c_squ * c_squ) - u_squ / (2.0 * c_squ));

		
		//relaxation step
		f0[pos(x,y,ny)] = tf0[pos(x,y,ny)] + omega *
		       	(n_equ[0] - tf0[pos(x,y,ny)]);
		f1[pos(x,y,ny)] = tf1[pos(x,y,ny)] + omega *
		       	(n_equ[1] - tf1[pos(x,y,ny)]);
		f2[pos(x,y,ny)] = tf2[pos(x,y,ny)] + omega *
		       	(n_equ[2] - tf2[pos(x,y,ny)]);
		f3[pos(x,y,ny)] = tf3[pos(x,y,ny)] + omega *
		       	(n_equ[3] - tf3[pos(x,y,ny)]);
		f4[pos(x,y,ny)] = tf4[pos(x,y,ny)] + omega *
		       	(n_equ[4] - tf4[pos(x,y,ny)]);
		f5[pos(x,y,ny)] = tf5[pos(x,y,ny)] + omega *
		       	(n_equ[5] - tf5[pos(x,y,ny)]);
		f6[pos(x,y,ny)] = tf6[pos(x,y,ny)] + omega *
		       	(n_equ[6] - tf6[pos(x,y,ny)]);
		f7[pos(x,y,ny)] = tf7[pos(x,y,ny)] + omega *
		       	(n_equ[7] - tf7[pos(x,y,ny)]);
		//for (i = 0; i < l->n; i++) {
		//	l->node[x][y][i] = l->temp[x][y][i] + omega * (n_equ[i] - l->temp[x][y][i]);
		//}	
	}
}

void lb::relaxation( void )
{
	/* here a kernel call */
	dim3 threads( BLOCK_SIZE, BLOCK_SIZE );
	dim3 grid( (nx+BLOCK_SIZE-1)/threads.x, (ny+BLOCK_SIZE-1)/threads.y );
	relaxation_kernel<<< grid, threads >>>(
		thrust::raw_pointer_cast(&d_f0[0]),
		thrust::raw_pointer_cast(&d_f1[0]),
		thrust::raw_pointer_cast(&d_f2[0]),
		thrust::raw_pointer_cast(&d_f3[0]),
		thrust::raw_pointer_cast(&d_f4[0]),
		thrust::raw_pointer_cast(&d_f5[0]),
		thrust::raw_pointer_cast(&d_f6[0]),
		thrust::raw_pointer_cast(&d_f7[0]),
		thrust::raw_pointer_cast(&d_f8[0]),
		// temps from here
		thrust::raw_pointer_cast(&d_tf0[0]),
		thrust::raw_pointer_cast(&d_tf1[0]),
		thrust::raw_pointer_cast(&d_tf2[0]),
		thrust::raw_pointer_cast(&d_tf3[0]),
		thrust::raw_pointer_cast(&d_tf4[0]),
		thrust::raw_pointer_cast(&d_tf5[0]),
		thrust::raw_pointer_cast(&d_tf6[0]),
		thrust::raw_pointer_cast(&d_tf7[0]),
		thrust::raw_pointer_cast(&d_tf8[0]),
		// others
		thrust::raw_pointer_cast(&d_obst[0]),
		nx, ny, omega );
}
