
#include <hip/hip_runtime.h>

#define NUM_SMS (24)
#define NUM_THREADS_PER_SM (384)
#define NUM_THREADS_PER_BLOCK (192)
#define NUM_BLOCKS ((NUM_THREADS_PER_SM / NUM_THREADS_PER_BLOCK) * NUM_SMS)
#define NUM_ITERATIONS 99999
 
// 128 MAD instructions
#define FMAD128(a, b) \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
 
//__shared__ float result[NUM_THREADS_PER_BLOCK];
extern __shared__ float result[];
 
__global__ void gflops( unsigned int n, float v, clock_t *timer )
{
	int idx= blockIdx.x * blockDim.x + threadIdx.x;
	__threadfence();
	if( idx == 0 )
		timer[0]= clock();

	float a = result[idx];  // this ensures the mads don't get compiled out
	float b = 1.01f;

	for (int i = 0; i < n; i++)
	{
		FMAD128(a, b);
		FMAD128(a, b);
		FMAD128(a, b);
		FMAD128(a, b);
		FMAD128(a, b);
		FMAD128(a, b);
		FMAD128(a, b);
		FMAD128(a, b);
		FMAD128(a, b);
		FMAD128(a, b);
		FMAD128(a, b);
		FMAD128(a, b);
		FMAD128(a, b);
		FMAD128(a, b);
		FMAD128(a, b);
		FMAD128(a, b);
	}
	result[idx] = a + b;

	__threadfence();
	if( idx == 0 )
		timer[1]= clock();
}
