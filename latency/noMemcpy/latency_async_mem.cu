 
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
 
#ifndef DEVICE
#define DEVICE 0
#endif

#include "cuda_safe.h"
#include "kernel_clock_timed.cu"
 
int
main(int argc, char** argv)
{
	clock_t *d_timer, *h_timer;
	struct timeval t0, t1, t2;
	float time1_0, time2_0;
	float k_time;
	int max_work=1000;
	double max_time= 100*1e3;
	unsigned long mem_size, mem_size_clock, shared_mem_size;
	int i, nmax=100;
	unsigned int sm= 30, thread= 128;
	hipStream_t stream1;
        hipDeviceProp_t deviceProp;
	float *d_data;
	unsigned int mem_size_result;
	unsigned int N;
	unsigned int offset= 0;

	if( argc > 1 )
		max_work= atol( argv[1] );

	CUDA_SAFE_CALL( hipSetDevice(DEVICE) );
	shared_mem_size= 0;

	mem_size_result= sm * thread * sizeof(float);
	N= sm * thread;
	hipMalloc( (void**)&d_data, mem_size_result );

	hipStreamCreate( &stream1 );

	mem_size_clock= sizeof(clock_t) * 2;
	CUDA_SAFE_CALL( hipHostAlloc( &h_timer, mem_size_clock,
				hipHostMallocDefault ) );
	hipMalloc( (void**)&d_timer, mem_size_clock );
        hipGetDeviceProperties(&deviceProp, DEVICE);

	gflops_heavy<<<sm, thread, shared_mem_size, stream1>>>(max_work,
			d_data, N, offset, d_timer);
	CUDA_SAFE_THREAD_SYNC();
	CUDA_SAFE_CALL( hipDeviceSynchronize() );
	/* search for some work near to 100 ms */
	i= max_work;
	do {
		do {
		gflops_heavy<<<sm, thread, shared_mem_size, stream1>>>(i,
			d_data, N, offset, d_timer);
		CUDA_SAFE_CALL( hipStreamSynchronize(stream1) );
		CUDA_SAFE_CALL( hipMemcpy( h_timer, d_timer, mem_size_clock,
			hipMemcpyDeviceToHost) );
		k_time= 1e6*((h_timer[1]-h_timer[0])/(deviceProp.clockRate*1e3f));
		} while( k_time < 0 );
		i+=10;
	} while( k_time < max_time );
	max_work= i-10;

	fprintf( stdout, "# max_work=%d time=%.2f \n", max_work, k_time );
	fprintf( stdout, "# size(B) t1-t0 t2-t0 time_kernel\n" );
	fflush(stdout);

	time1_0= time2_0= k_time= 0;

	for( i= 0; i < nmax; i++ ){
		gettimeofday( &t0, 0 );
		gflops_heavy<<<sm, thread, shared_mem_size, stream1>>>(max_work,
			       d_data, N, offset, d_timer);
		CUDA_SAFE_THREAD_SYNC();
		gettimeofday( &t1, 0 );
		CUDA_SAFE_CALL( hipStreamSynchronize(stream1) );
		gettimeofday( &t2, 0 );
		time1_0= (t1.tv_sec-t0.tv_sec)*1e6+(t1.tv_usec-t0.tv_usec);
		time2_0= (t2.tv_sec-t0.tv_sec)*1e6+(t2.tv_usec-t0.tv_usec);
		CUDA_SAFE_CALL( hipMemcpy( h_timer, d_timer, mem_size_clock,
				hipMemcpyDeviceToHost) );
		k_time= 1e6*((h_timer[1]-h_timer[0])/(deviceProp.clockRate*1e3f));
		if(  k_time > 0 ){
			fprintf( stdout, "%10u %10.2f %10.2f %10.2f\n",
				mem_size,
				time1_0, time2_0, k_time );
		} else {
			fprintf( stdout, "# %10u %10.2f %10.2f %10.2f\n",
				mem_size,
				time1_0, time2_0, k_time );
		}
	}
	fflush(stdout);
	hipFree( d_data );
	hipHostFree( h_timer );
	hipFree( d_timer );
	hipDeviceReset();
}
