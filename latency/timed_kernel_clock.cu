#include "hip/hip_runtime.h"
 
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
 
#ifndef DEVICE
#define DEVICE 0
#endif

#include "cuda_safe.h"

#define NUM_SMS (24)
#define NUM_THREADS_PER_SM (384)
#define NUM_THREADS_PER_BLOCK (192)
#define NUM_BLOCKS ((NUM_THREADS_PER_SM / NUM_THREADS_PER_BLOCK) * NUM_SMS)
#define NUM_ITERATIONS 99999
 
// 128 MAD instructions
#define FMAD128(a, b) \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
     a = b * a + b; \
     b = a * b + a; \
 
__shared__ float result[NUM_THREADS_PER_BLOCK];
 
__global__ void gflops( unsigned int n, clock_t *timer )
{
	__threadfence();
	if( threadIdx.x == 0 )
		timer[0]= clock();

   float a = result[threadIdx.x];  // this ensures the mads don't get compiled out
   float b = 1.01f;
 
   for (int i = 0; i < n; i++)
   {
       FMAD128(a, b);
       FMAD128(a, b);
       FMAD128(a, b);
       FMAD128(a, b);
       FMAD128(a, b);
       FMAD128(a, b);
       FMAD128(a, b);
       FMAD128(a, b);
       FMAD128(a, b);
       FMAD128(a, b);
       FMAD128(a, b);
       FMAD128(a, b);
       FMAD128(a, b);
       FMAD128(a, b);
       FMAD128(a, b);
       FMAD128(a, b);
   }
   result[threadIdx.x] = a + b;

	__threadfence();
	if( threadIdx.x == 0 )
		timer[1]= clock();
}
 
int
main(int argc, char** argv)
{
	int max_work=1000;
	unsigned int mem_size;
	clock_t *d_timer, *h_timer;
        hipDeviceProp_t deviceProp;
	float sys_time;
	float uclock;
	int i, max_iter= 100;
	struct timeval t0, t1;

	CUDA_SAFE_CALL( hipSetDevice(DEVICE) );
	mem_size= sizeof(clock_t) * 2;
	h_timer= (clock_t *) malloc(mem_size);
	hipMalloc( (void**)&d_timer, mem_size );
        hipGetDeviceProperties(&deviceProp, DEVICE);

	gflops<<<1, 1>>>(max_work, d_timer);
	CUDA_SAFE_CALL( hipDeviceSynchronize() );
	fprintf( stdout, "# clock(us) cpu(us) diff(us)\n" );
	fflush( stdout );

	for( i= 0; i < max_iter; i++ ) {
		CUDA_SAFE_CALL( hipDeviceSynchronize() );
		gettimeofday( &t0, 0 );
		gflops<<<1, 1>>>(max_work, d_timer);
		CUDA_SAFE_CALL( hipDeviceSynchronize() );
		gettimeofday( &t1, 0 );
		sys_time= (t1.tv_sec-t0.tv_sec)*1e6+(t1.tv_usec-t0.tv_usec);
		hipMemcpy( h_timer, d_timer, mem_size, hipMemcpyDeviceToHost);
		if( (h_timer[1]-h_timer[0]) > 0 ){
			uclock= (h_timer[1]-h_timer[0])/(deviceProp.clockRate*1e3f);
			fprintf( stdout, "%.2f %.2f %.2f\n", uclock*1e6,
					sys_time,
				sys_time-uclock*1e6 );
			fflush(stdout);
		}
	}
	hipDeviceReset();
}
