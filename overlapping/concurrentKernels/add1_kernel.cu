
#include <hip/hip_runtime.h>

#define CUDA_SAFE_CALL(call) do {                                 \
    hipError_t err = call;                                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
        exit(EXIT_FAILURE);                                                  \
    }                                                                        \
    }while(0)


#define	NTASKS	8
#define MAX_MEM		25	/* 32MB */
#define GRID_SIZE	64
#define	BLOCK_SIZE	512

__global__ void add1( float* array, unsigned int size )
{
  const unsigned int per_thread = size / (gridDim.x * blockDim.x);
  unsigned int i = (blockIdx.x*blockDim.x + threadIdx.x) * per_thread;

  unsigned int j = size;
  if ( (blockIdx.x*blockDim.x + threadIdx.x) != (gridDim.x * blockDim.x-1) )
	  j = i + per_thread;

  for (; i < j; ++i)
	  ++array[i];
}

int check( const float *data, const unsigned int n, const float v )
{
	for( int i= 0; i < n; i++ )
		if( data[i] != v )
			return 1;

	return 0;
}

