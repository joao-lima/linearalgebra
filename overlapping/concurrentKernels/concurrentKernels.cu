#include <stdio.h>

#define	NSTREAMS	4
#include "add1_kernel.cu"

int main(int argc, char **argv)
{
    int cuda_device = 0;
    unsigned int mem_size = (1 << 26);
    unsigned int nstreams = NSTREAMS;
    unsigned int ntasks = NTASKS;
    unsigned int nevents = ntasks * 2;
    float *h_data[NTASKS], *d_data[NTASKS];
    float elapsed_time= 0;

    cuda_device = 0;
    hipDeviceProp_t deviceProp;
    CUDA_SAFE_CALL( hipGetDevice(&cuda_device));	

    CUDA_SAFE_CALL( hipGetDeviceProperties(&deviceProp, cuda_device) );
    if( (deviceProp.concurrentKernels == 0 ))
        printf("> GPU does not support concurrent kernel execution, kernel runs will be serialized\n");

    //printf("> Detected Compute SM %d.%d hardware with %d multi-processors\n", deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount); 

    hipStream_t *streams = (hipStream_t*) malloc(nstreams * sizeof(hipStream_t));
    hipEvent_t *events = (hipEvent_t*) malloc(nevents * sizeof(hipEvent_t));
    for(int i = 0; i < nstreams; i++)
	CUDA_SAFE_CALL( hipStreamCreate(&(streams[i])) );
    for(int i = 0; i < nevents; i++)
        CUDA_SAFE_CALL( hipEventCreateWithFlags(&(events[i]), hipEventDisableTiming) );

    for( int i= 0; i < ntasks; i++ ) {
	CUDA_SAFE_CALL( hipHostMalloc((void**)&h_data[i], mem_size) ); 
	CUDA_SAFE_CALL( hipMalloc((void**)&d_data[i], mem_size) );
	for( int j= 0; j < (mem_size/sizeof(float)); j++ )
		h_data[i][j]= 1.0f;
    }
    // create CUDA event handles
    hipEvent_t start_event, stop_event;
    CUDA_SAFE_CALL( hipEventCreate(&start_event) );
    CUDA_SAFE_CALL( hipEventCreate(&stop_event) );
	
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    hipEventRecord(start_event, 0);
    // queue nkernels in separate streams and record when they are done
    for( int i=0; i < ntasks; ++i) {
	CUDA_SAFE_CALL( hipMemcpyAsync( d_data[i], h_data[i], mem_size,
			hipMemcpyHostToDevice, streams[0] ));
	CUDA_SAFE_CALL( hipEventRecord( events[i*2], streams[0] ) );

	CUDA_SAFE_CALL( hipStreamWaitEvent( streams[1], events[i*2], 0) );
        add1<<<1,256,0,streams[1]>>>(d_data[i], (mem_size/sizeof(float)) );
	CUDA_SAFE_CALL( hipEventRecord( events[i*2+1], streams[1] ) );

	CUDA_SAFE_CALL( hipStreamWaitEvent( streams[2], events[i*2+1], 0) );
	CUDA_SAFE_CALL( hipMemcpyAsync( h_data[i], d_data[i], mem_size,
			hipMemcpyDeviceToHost, streams[2] ) );
    }

    // in this sample we just wait until the GPU is done
    CUDA_SAFE_CALL( hipEventRecord(stop_event, 0) );
    CUDA_SAFE_CALL( hipEventSynchronize(stop_event) );
    CUDA_SAFE_CALL( hipEventElapsedTime(&elapsed_time,
		    start_event, stop_event) );
    
    printf("Measured time for sample = %.4f\n", elapsed_time);

    for( int i= 0; i < ntasks; i++ )
	    if( check( h_data[i], mem_size/sizeof(float), 11) )
		    fprintf(stdout, "ERROR at task %d\n", i ); fflush(stdout);
    
    // release resources
    for(int i = 0; i < nstreams; i++)
		hipStreamDestroy(streams[i]);

    for(int i = 0; i < nevents; i++)
		hipEventDestroy(events[i]);
    for( int i= 0; i < ntasks; i++ ) {
	    hipHostFree(h_data[i]);
	    hipFree(d_data[i]);
    }

    free(streams);
    free(events);
    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);

    hipDeviceReset();
}
