#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

//
// This sample demonstrates the use of streams for concurrent execution. It also illustrates how to 
// introduce dependencies between CUDA streams with the new hipStreamWaitEvent function introduced 
// in CUDA 3.2.
//
// Devices of compute capability 1.x will run the kernels one after another
// Devices of compute capability 2.0 or higher can overlap the kernels
//

#include <stdio.h>

const char *sSDKsample = "concurrentKernels";

#define CUDA_SAFE_CALL(call) do {                                 \
    hipError_t err = call;                                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
        exit(EXIT_FAILURE);                                                  \
    }                                                                        \
    }while(0)


#define	NTASKS	4
#define	NSTREAMS	4

__global__ void add1( float* array, unsigned int size )
{
  const unsigned int per_thread = size / blockDim.x;
  unsigned int i = threadIdx.x * per_thread;

  unsigned int j = size;
  if (threadIdx.x != (blockDim.x - 1)) j = i + per_thread;

  for (; i < j; ++i) ++array[i];
}

int check( const float *data, const unsigned int n, const float v )
{
	for( int i= 0; i < n; i++ )
		if( data[i] != v )
			return 1;

	return 0;
}

int main(int argc, char **argv)
{
    int cuda_device = 0;
    unsigned int mem_size = (1 << 27);
    unsigned int nstreams = NSTREAMS;
    unsigned int ntasks = NTASKS;
    unsigned int nevents = ntasks * 2;
    float *h_data[NTASKS], *d_data[NTASKS];
    float elapsed_time= 0;

    printf("[ %s ]\n", sSDKsample);
    cuda_device = 0;
    hipDeviceProp_t deviceProp;
    CUDA_SAFE_CALL( hipGetDevice(&cuda_device));	

    CUDA_SAFE_CALL( hipGetDeviceProperties(&deviceProp, cuda_device) );
    if( (deviceProp.concurrentKernels == 0 ))
        printf("> GPU does not support concurrent kernel execution, kernel runs will be serialized\n");

    printf("> Detected Compute SM %d.%d hardware with %d multi-processors\n", deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount); 

    hipStream_t *streams = (hipStream_t*) malloc(nstreams * sizeof(hipStream_t));
    hipEvent_t *events = (hipEvent_t*) malloc(nevents * sizeof(hipEvent_t));
    for(int i = 0; i < nstreams; i++)
	CUDA_SAFE_CALL( hipStreamCreate(&(streams[i])) );
    for(int i = 0; i < nevents; i++)
        CUDA_SAFE_CALL( hipEventCreateWithFlags(&(events[i]), hipEventDisableTiming) );

    for( int i= 0; i < ntasks; i++ ) {
	CUDA_SAFE_CALL( hipHostMalloc((void**)&h_data[i], mem_size) ); 
	CUDA_SAFE_CALL( hipMalloc((void**)&d_data[i], mem_size) );
	for( int j= 0; j < (mem_size/sizeof(float)); j++ )
		h_data[i][j]= 1.0f;
    }
    // create CUDA event handles
    hipEvent_t start_event, stop_event;
    CUDA_SAFE_CALL( hipEventCreate(&start_event) );
    CUDA_SAFE_CALL( hipEventCreate(&stop_event) );
	
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    hipEventRecord(start_event, 0);
    // queue nkernels in separate streams and record when they are done
    for( int i=0; i < ntasks; ++i) {
	CUDA_SAFE_CALL( hipMemcpyAsync( d_data[i], h_data[i], mem_size,
			hipMemcpyHostToDevice, streams[0] ));
	CUDA_SAFE_CALL( hipEventRecord( events[i*2], streams[0] ) );

	CUDA_SAFE_CALL( hipStreamWaitEvent( streams[1], events[i*2], 0) );
        add1<<<1,256,0,streams[1]>>>(d_data[i], (mem_size/sizeof(float)) );
	CUDA_SAFE_CALL( hipEventRecord( events[i*2+1], streams[1] ) );

	CUDA_SAFE_CALL( hipStreamWaitEvent( streams[2], events[i*2+1], 0) );
	CUDA_SAFE_CALL( hipMemcpyAsync( h_data[i], d_data[i], mem_size,
			hipMemcpyDeviceToHost, streams[2] ) );
    }

    // in this sample we just wait until the GPU is done
    CUDA_SAFE_CALL( hipEventRecord(stop_event, 0) );
    CUDA_SAFE_CALL( hipEventSynchronize(stop_event) );
    CUDA_SAFE_CALL( hipEventElapsedTime(&elapsed_time,
		    start_event, stop_event) );
    
    printf("Measured time for sample = %.3fs\n", elapsed_time/1000.0f);

    for( int i= 0; i < ntasks; i++ )
	    if( check( h_data[i], mem_size/sizeof(float), 2.0) )
		    fprintf(stdout, "ERROR at task %d\n", i ); fflush(stdout);
    
    // release resources
    for(int i = 0; i < nstreams; i++)
		hipStreamDestroy(streams[i]);

    for(int i = 0; i < nevents; i++)
		hipEventDestroy(events[i]);
    for( int i= 0; i < ntasks; i++ ) {
	    hipHostFree(h_data[i]);
	    hipFree(d_data[i]);
    }

    free(streams);
    free(events);
    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);

    hipDeviceReset();
}
