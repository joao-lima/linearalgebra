#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>

const char *sSDKsample = "concurrentKernels";

#define CUDA_SAFE_CALL(call) do {                                 \
    hipError_t err = call;                                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %d.\n",        \
                __FILE__, __LINE__, err );              \
        exit(EXIT_FAILURE);                                                  \
    }                                                                        \
    }while(0)


#define	NTASKS	2

__global__ void add1( float* array, unsigned int size )
{
  const unsigned int per_thread = size / blockDim.x;
  unsigned int i = threadIdx.x * per_thread;

  unsigned int j = size;
  if (threadIdx.x != (blockDim.x - 1)) j = i + per_thread;

  unsigned int k;
  for (; i < j; ++i)
  for(k = 0; k < 10;k++)
	  ++array[i];
}

int check( const float *data, const unsigned int n, const float v )
{
	for( int i= 0; i < n; i++ )
		if( data[i] != v )
			return 1;

	return 0;
}

int main(int argc, char **argv)
{
    int cuda_device = 0;
    unsigned int mem_size = (1 << 28);
    unsigned int ntasks = NTASKS;
    float *h_data[NTASKS];
    hipDeviceptr_t d_data[NTASKS];
    float elapsed_time= 0;
    unsigned int nmax= 100;

    hipInit(0);

    hipDevice_t cuDevice;
    hipCtx_t cuContext;
    unsigned int flags = hipDeviceScheduleYield;
    cuda_device = 0;
    CUDA_SAFE_CALL( hipDeviceGet(&cuDevice, cuda_device) );	
    CUDA_SAFE_CALL( hipCtxCreate( &cuContext, flags, cuDevice ) );

    for( int i= 0; i < ntasks; i++ ) {
	CUDA_SAFE_CALL( hipHostAlloc( (void**)&h_data[i], mem_size,
			hipHostMallocPortable) ); 
	CUDA_SAFE_CALL( hipMalloc( &d_data[i], mem_size) );
	//CUDA_SAFE_CALL( hipMemsetD16( d_data[i], 0, mem_size ) );
	for( int j= 0; j < (mem_size/sizeof(float)); j++ )
		h_data[i][j]= 1.0f;
    }
    // create CUDA event handles
    hipEvent_t  start_event, stop_event;
    CUDA_SAFE_CALL( hipEventCreateWithFlags(&start_event, hipEventDefault) );
    CUDA_SAFE_CALL( hipEventCreateWithFlags(&stop_event, hipEventDefault) );
    hipStream_t s1, s2;
    CUDA_SAFE_CALL( hipStreamCreateWithFlags(&s1, 0) );
    CUDA_SAFE_CALL( hipStreamCreateWithFlags(&s2, 0) );

    unsigned int i= 0;
    CUDA_SAFE_CALL( hipCtxSynchronize() );
    hipEventRecord(start_event, 0);
    for(i= 0; i < nmax; i++) {
	    CUDA_SAFE_CALL( hipMemcpyHtoDAsync( d_data[0], h_data[0], mem_size,
			s1 ));
	    CUDA_SAFE_CALL( hipMemcpyDtoHAsync( h_data[1], d_data[1], mem_size,
			   s2) );
	    //hipCtxSynchronize();
    }
    // in this sample we just wait until the GPU is done
    CUDA_SAFE_CALL( hipEventRecord(stop_event, 0) );
    CUDA_SAFE_CALL( hipEventSynchronize(stop_event) );
    CUDA_SAFE_CALL( hipEventElapsedTime(&elapsed_time, start_event, stop_event) );
    printf("Time duplex = %.4f ms\n", elapsed_time/nmax);

    CUDA_SAFE_CALL( hipCtxSynchronize() );
    hipEventRecord(start_event, 0);
    for(i= 0; i < nmax; i++) {
	    CUDA_SAFE_CALL( hipMemcpyHtoDAsync( d_data[0], h_data[0], mem_size,
			s1 ));
	    //hipCtxSynchronize();
    }
    // in this sample we just wait until the GPU is done
    CUDA_SAFE_CALL( hipEventRecord(stop_event, 0) );
    CUDA_SAFE_CALL( hipEventSynchronize(stop_event) );
    CUDA_SAFE_CALL( hipEventElapsedTime(&elapsed_time, start_event, stop_event) );
    printf("Time simple = %.4f ms\n", elapsed_time/nmax);

    for( int i= 0; i < ntasks; i++ ) {
	    hipHostFree(h_data[i]);
	    hipFree(d_data[i]);
    }

    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);
    hipStreamDestroy(s1);
    hipStreamDestroy(s2);
    hipCtxPopCurrent(&cuContext);
    return 0;
}
