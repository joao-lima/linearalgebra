#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

//
// This sample demonstrates the use of streams for concurrent execution. It also illustrates how to 
// introduce dependencies between CUDA streams with the new hipStreamWaitEvent function introduced 
// in CUDA 3.2.
//
// Devices of compute capability 1.x will run the kernels one after another
// Devices of compute capability 2.0 or higher can overlap the kernels
//

#include <stdio.h>

const char *sSDKsample = "concurrentKernels";

#define CUDA_SAFE_CALL(call) do {                                 \
    hipError_t err = call;                                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
        exit(EXIT_FAILURE);                                                  \
    }                                                                        \
    }while(0)


#define	NTASKS	32

__global__ void add1( float* array, unsigned int size )
{
  const unsigned int per_thread = size / blockDim.x;
  unsigned int i = threadIdx.x * per_thread;

  unsigned int j = size;
  if (threadIdx.x != (blockDim.x - 1)) j = i + per_thread;

  unsigned int k;
  for (; i < j; ++i)
  for(k = 0; k < 10;k++)
	  ++array[i];
}

int check( const float *data, const unsigned int n, const float v )
{
	for( int i= 0; i < n; i++ )
		if( data[i] != v )
			return 1;

	return 0;
}

int main(int argc, char **argv)
{
    int cuda_device = 0;
    unsigned int mem_size = (1 << 26);
    unsigned int ntasks = NTASKS;
    unsigned int itasks = 0; // n of ready tasks
    float *h_data[NTASKS], *d_data[NTASKS];
    float elapsed_time= 0;

    cuda_device = 0;
    hipDeviceProp_t deviceProp;
    CUDA_SAFE_CALL( hipGetDevice(&cuda_device));	

    CUDA_SAFE_CALL( hipGetDeviceProperties(&deviceProp, cuda_device) );
    if( (deviceProp.concurrentKernels == 0 ))
        printf("> GPU does not support concurrent kernel execution, kernel runs will be serialized\n");

    //printf("> Detected Compute SM %d.%d hardware with %d multi-processors\n", deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount); 

    hipStream_t *streams = (hipStream_t*) malloc(ntasks * sizeof(hipStream_t));
    for(int i = 0; i < ntasks; i++)
	CUDA_SAFE_CALL( hipStreamCreate(&(streams[i])) );

    for( int i= 0; i < ntasks; i++ ) {
	CUDA_SAFE_CALL( hipHostMalloc((void**)&h_data[i], mem_size) ); 
	CUDA_SAFE_CALL( hipMalloc((void**)&d_data[i], mem_size) );
	for( int j= 0; j < (mem_size/sizeof(float)); j++ )
		h_data[i][j]= 1.0f;
    }
    // create CUDA event handles
    hipEvent_t start_event, stop_event;
    CUDA_SAFE_CALL( hipEventCreate(&start_event) );
    CUDA_SAFE_CALL( hipEventCreate(&stop_event) );
	
    hipStream_t stream_HtoD, stream_DtoH, stream_k;
    CUDA_SAFE_CALL( hipStreamCreate(&stream_HtoD) );
    CUDA_SAFE_CALL( hipStreamCreate(&stream_DtoH) );
    CUDA_SAFE_CALL( hipStreamCreate(&stream_k) );

    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    hipEventRecord(start_event, 0);

    unsigned int i_HtoD= 0, i_DtoH= 0, i_k= 0;

  CUDA_SAFE_CALL( hipMemcpyAsync( d_data[i_HtoD],
	h_data[i_HtoD], mem_size,
	hipMemcpyHostToDevice, stream_HtoD ));
	   i_HtoD++;
	 hipStreamSynchronize( stream_HtoD );
		  fprintf(stdout,"kernel k=%d\n",i_k);fflush(stdout);
       add1<<<1,256,0,stream_k>>>(d_data[i_k], (mem_size/sizeof(float)) );
       i_k++;
    while( itasks < ntasks ) {
	if( hipStreamQuery( stream_k ) == hipErrorNotReady ) {
	       if( i_HtoD < ntasks &&
			hipStreamQuery( stream_HtoD )  == hipSuccess ) {
		  fprintf(stdout,"HtoD k=%d\n",i_HtoD);fflush(stdout);
		  CUDA_SAFE_CALL( hipMemcpyAsync( d_data[i_HtoD],
			h_data[i_HtoD], mem_size,
			hipMemcpyHostToDevice, stream_HtoD ));
		   i_HtoD++;
	       }
	       hipStreamSynchronize( stream_k ) ;
	       continue;
	} else {
	  fprintf(stdout,"DtoH for k=%d\n",i_DtoH);fflush(stdout);
	CUDA_SAFE_CALL( hipMemcpyAsync( h_data[i_DtoH],
		d_data[i_DtoH], mem_size,
		hipMemcpyDeviceToHost, stream_DtoH ) );
	i_DtoH++;
	itasks++;
	if( i_k < ntasks ) {
	  fprintf(stdout,"kernel k=%d\n",i_k);fflush(stdout);
        add1<<<1,256,0,stream_k>>>(d_data[i_k], (mem_size/sizeof(float)) );
       i_k++;
	}
       if( i_HtoD < ntasks ) {
	  fprintf(stdout,"HtoD k=%d\n",i_HtoD);fflush(stdout);
	  CUDA_SAFE_CALL( hipMemcpyAsync( d_data[i_HtoD],
		h_data[i_HtoD], mem_size,
		hipMemcpyHostToDevice, stream_HtoD ));
	   i_HtoD++;
       }

	}
    }

    // in this sample we just wait until the GPU is done
    CUDA_SAFE_CALL( hipEventRecord(stop_event, 0) );
    CUDA_SAFE_CALL( hipEventSynchronize(stop_event) );
    CUDA_SAFE_CALL( hipEventElapsedTime(&elapsed_time,
		    start_event, stop_event) );
    
    printf("Measured time for sample = %.3fs\n", elapsed_time/1000.0f);

    for( int i= 0; i < ntasks; i++ )
	    if( check( h_data[i], mem_size/sizeof(float), 11) )
		    fprintf(stdout, "ERROR at task %d\n", i ); fflush(stdout);
    
    // release resources
    for(int i = 0; i < ntasks; i++)
		hipStreamDestroy(streams[i]);

    for( int i= 0; i < ntasks; i++ ) {
	    hipHostFree(h_data[i]);
	    hipFree(d_data[i]);
    }

    free(streams);
    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);

    hipDeviceReset();
}
