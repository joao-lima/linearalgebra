#include <stdio.h>

#include "add1_kernel.cu"

int main(int argc, char **argv)
{
    int cuda_device = 0;
    unsigned int mem_size = (1 << MAX_MEM);
    unsigned int ntasks = NTASKS;
    float *h_data[NTASKS], *d_data[NTASKS];
    float elapsed_time= 0;

    cuda_device = 0;
    hipDeviceProp_t deviceProp;
    CUDA_SAFE_CALL( hipGetDevice(&cuda_device));	

    CUDA_SAFE_CALL( hipGetDeviceProperties(&deviceProp, cuda_device) );
    if( (deviceProp.concurrentKernels == 0 ))
        printf("> GPU does not support concurrent kernel execution, kernel runs will be serialized\n");

    //printf("> Detected Compute SM %d.%d hardware with %d multi-processors\n", deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount); 

    hipStream_t *streams = (hipStream_t*) malloc(ntasks * sizeof(hipStream_t));
    for(int i = 0; i < ntasks; i++)
	CUDA_SAFE_CALL( hipStreamCreate(&(streams[i])) );

    for( int i= 0; i < ntasks; i++ ) {
	CUDA_SAFE_CALL( hipHostMalloc((void**)&h_data[i], mem_size) ); 
	CUDA_SAFE_CALL( hipMalloc((void**)&d_data[i], mem_size) );
	for( int j= 0; j < (mem_size/sizeof(float)); j++ )
		h_data[i][j]= 1.0f;
    }
    // create CUDA event handles
    hipEvent_t start_event, stop_event;
    CUDA_SAFE_CALL( hipEventCreate(&start_event) );
    CUDA_SAFE_CALL( hipEventCreate(&stop_event) );
	
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    hipEventRecord(start_event, 0);
    // queue nkernels in separate streams and record when they are done
    for( int i=0; i < ntasks; ++i) {
	CUDA_SAFE_CALL( hipMemcpyAsync( d_data[i], h_data[i], mem_size,
			hipMemcpyHostToDevice, streams[i] ));

        add1<<<GRID_SIZE,BLOCK_SIZE,0,streams[i]>>>(d_data[i], (mem_size/sizeof(float)) );

	CUDA_SAFE_CALL( hipMemcpyAsync( h_data[i], d_data[i], mem_size,
			hipMemcpyDeviceToHost, streams[i] ) );
    }

    // in this sample we just wait until the GPU is done
    CUDA_SAFE_CALL( hipEventRecord(stop_event, 0) );
    CUDA_SAFE_CALL( hipEventSynchronize(stop_event) );
    CUDA_SAFE_CALL( hipEventElapsedTime(&elapsed_time,
		    start_event, stop_event) );
    
    printf("Measured time for sample = %.4f\n", elapsed_time);

    for( int i= 0; i < ntasks; i++ )
	    if( check( h_data[i], mem_size/sizeof(float), 2) )
		    fprintf(stdout, "ERROR at task %d\n", i ); fflush(stdout);
    
    // release resources
    for(int i = 0; i < ntasks; i++)
		hipStreamDestroy(streams[i]);

    for( int i= 0; i < ntasks; i++ ) {
	    hipHostFree(h_data[i]);
	    hipFree(d_data[i]);
    }

    free(streams);
    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);

    hipDeviceReset();
}
