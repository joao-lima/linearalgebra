#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

//
// This sample demonstrates the use of streams for concurrent execution. It also illustrates how to 
// introduce dependencies between CUDA streams with the new hipStreamWaitEvent function introduced 
// in CUDA 3.2.
//
// Devices of compute capability 1.x will run the kernels one after another
// Devices of compute capability 2.0 or higher can overlap the kernels
//

#include <stdio.h>

const char *sSDKsample = "concurrentKernels";

#define CUDA_SAFE_CALL(call) do {                                 \
    hipError_t err = call;                                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
        exit(EXIT_FAILURE);                                                  \
    }                                                                        \
    }while(0)


#define	NTASKS	2
#define	NSTREAMS	4
#define BLOCK_SIZE	256

__global__ void add1( float* array, unsigned int size )
{
  const unsigned int per_thread = size / blockDim.x;
  unsigned int i = threadIdx.x * per_thread;
  extern __shared__ int a[];

  unsigned int j = size;
  if (threadIdx.x != (blockDim.x - 1)) j = i + per_thread;

  unsigned int k;
  a[threadIdx.x]=1;
//  for (; i < j; ++i)
//	  a[i]=1;
  for(k = 0; k < 100;k++)
	  a[threadIdx.x]++;
//		  ++array[i];
}

int check( const float *data, const unsigned int n, const float v )
{
	for( int i= 0; i < n; i++ )
		if( data[i] != v )
			return 1;

	return 0;
}

int main(int argc, char **argv)
{
    int cuda_device = 0;
    unsigned int mem_size = (1 << 26);
    unsigned int nstreams = NTASKS;
    unsigned int ntasks = NTASKS;
    unsigned int nevents = ntasks * 2;
    float *h_data[NTASKS], *d_data[NTASKS];
    float elapsed_time= 0;

    cuda_device = 0;
    hipDeviceProp_t deviceProp;
    CUDA_SAFE_CALL( hipGetDevice(&cuda_device));	

    CUDA_SAFE_CALL( hipGetDeviceProperties(&deviceProp, cuda_device) );
    if( (deviceProp.concurrentKernels == 0 ))
        printf("> GPU does not support concurrent kernel execution, kernel runs will be serialized\n");

    //printf("> Detected Compute SM %d.%d hardware with %d multi-processors\n", deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount); 

    hipStream_t *streams = (hipStream_t*) malloc(nstreams * sizeof(hipStream_t));
    hipEvent_t *events = (hipEvent_t*) malloc(nevents * sizeof(hipEvent_t));
    for(int i = 0; i < nstreams; i++)
	CUDA_SAFE_CALL( hipStreamCreate(&(streams[i])) );
    for(int i = 0; i < nevents; i++)
        CUDA_SAFE_CALL( hipEventCreateWithFlags(&(events[i]), hipEventDisableTiming) );

    for( int i= 0; i < ntasks; i++ ) {
	CUDA_SAFE_CALL( hipHostMalloc((void**)&h_data[i], mem_size) ); 
	CUDA_SAFE_CALL( hipMalloc((void**)&d_data[i], mem_size) );
	for( int j= 0; j < (mem_size/sizeof(float)); j++ )
		h_data[i][j]= 1.0f;
    }
    // create CUDA event handles
    hipEvent_t start_event, stop_event;
    CUDA_SAFE_CALL( hipEventCreate(&start_event) );
    CUDA_SAFE_CALL( hipEventCreate(&stop_event) );

    hipStream_t DtoH;
    CUDA_SAFE_CALL( hipStreamCreate(&DtoH) );
	
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    hipEventRecord(start_event, 0);

    // queue nkernels in separate streams and record when they are done
    fprintf(stdout,"HtoD\n");fflush(stdout);
    for( int i=0; i < ntasks; ++i) {
	CUDA_SAFE_CALL( hipMemcpyAsync( d_data[i], h_data[i], mem_size,
			hipMemcpyHostToDevice, streams[i] ));
        add1<<<1,BLOCK_SIZE,BLOCK_SIZE*sizeof(int),streams[i]>>>(d_data[i], (mem_size/sizeof(float)) );
	CUDA_SAFE_CALL( hipEventRecord( events[i], streams[i] ) );
    }
    for( int i=0; i < ntasks; ++i) {
	CUDA_SAFE_CALL( hipStreamWaitEvent( DtoH, events[i], 0) );
	CUDA_SAFE_CALL( hipMemcpyAsync( h_data[i], d_data[i], mem_size,
			hipMemcpyDeviceToHost, DtoH ) );
    }
    fprintf(stdout,"sync\n");fflush(stdout);

    // in this sample we just wait until the GPU is done
    CUDA_SAFE_CALL( hipEventRecord(stop_event, 0) );
    CUDA_SAFE_CALL( hipEventSynchronize(stop_event) );
    CUDA_SAFE_CALL( hipEventElapsedTime(&elapsed_time,
		    start_event, stop_event) );
    
    printf("Measured time for sample = %.4f\n", elapsed_time);

    for( int i= 0; i < ntasks; i++ )
	    if( check( h_data[i], mem_size/sizeof(float), 11) )
		    fprintf(stdout, "ERROR at task %d\n", i ); fflush(stdout);
    
    // release resources
    for(int i = 0; i < nstreams; i++)
		hipStreamDestroy(streams[i]);
	hipStreamDestroy(DtoH);

    for(int i = 0; i < nevents; i++)
		hipEventDestroy(events[i]);
    for( int i= 0; i < ntasks; i++ ) {
	    hipHostFree(h_data[i]);
	    hipFree(d_data[i]);
    }

    free(streams);
    free(events);
    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);

    hipDeviceReset();
}
