/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation and 
 * any modifications thereto.  Any use, reproduction, disclosure, or distribution 
 * of this software and related documentation without an express license 
 * agreement from NVIDIA Corporation is strictly prohibited.
 * 
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <cutil_inline.h>

#include "add_kernel.cu"

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char** argv)
{
	unsigned int mem_size= (1 << 25);
	float elapsed_time_in_Ms= 0;
	float bandwidth_in_MBs= 0;
	int i, j, max_iter= 10;
	float *h_data, *d_data;
#define NSTREAM		2
	hipStream_t stream[NSTREAM];

	if( argc > 1 )
		mem_size =  (1 << atoi(argv[1]));

	unsigned int nelem= mem_size/sizeof(float);
	hipSetDevice( DEVICE );
	/* CUDA flags:
	hipHostMallocDefault, hipHostMallocPortable, hipHostMallocMapped,
	hipHostMallocWriteCombined */
	unsigned int flags= hipHostMallocDefault;
	// allocate host memory for matrices A and B
	CUDA_SAFE_CALL( hipHostAlloc( (void**)&h_data, mem_size, flags ) );
	for( i= 0; i < nelem; i++) h_data[i]= 1e0f;
	// allocate device memory
	CUDA_SAFE_CALL( hipMalloc((void**)&d_data, mem_size) );
	hipEvent_t e1, e2;
	hipEventCreate( &e1 );
	hipEventCreate( &e2 );
	for( j= 0; j < NSTREAM; j++ )
		hipStreamCreate( &stream[j] );
	// setup execution parameters
	dim3 threads( BLOCK_SIZE, 1 );
	dim3 grid( 128, 1);
	// number of elements per thread
	unsigned int nblock = nelem/(BLOCK_SIZE*grid.x);
	unsigned int n_per_stream = mem_size / NSTREAM;

	CUDA_SAFE_CALL( hipEventRecord( e1, 0 ) );
	for( i= 0; i < max_iter; i++ ){
		for( j= 0; j < NSTREAM; j++ ){
		CUDA_SAFE_CALL( hipMemcpyAsync( ((void*)d_data)+j*n_per_stream,
			((void*)h_data)+j*n_per_stream,
			n_per_stream, hipMemcpyHostToDevice, stream[j]) );
		}
		hipDeviceSynchronize();
		add_one<<< grid, threads >>>( d_data, nblock );
		CUDA_SAFE_CALL( hipMemcpy( h_data, d_data, mem_size,
				      hipMemcpyDeviceToHost) );
	}
	CUDA_SAFE_CALL( hipEventRecord( e2, 0 ) );
	CUDA_SAFE_CALL( hipEventSynchronize( e2 ) );
	CUDA_SAFE_CALL( hipEventElapsedTime( &elapsed_time_in_Ms, e1, e2 ) );
	bandwidth_in_MBs= 1e3f * max_iter * (mem_size * 2.0f) / 
	       	(elapsed_time_in_Ms * (float)(1 << 20));
	fprintf( stdout, "pinned_async size(MB)= %9u time(s)= %.3f bandwidth(MB/s)= %.1f\n",
		mem_size/(1<<20), elapsed_time_in_Ms/(1e3f*max_iter),
	       	bandwidth_in_MBs );

	if( check( h_data, 11e0f, nelem) == 0 )
		fprintf( stdout, "test FAILED\n" );

	// clean up memory
	CUDA_SAFE_CALL( hipEventDestroy( e1 ) );
	CUDA_SAFE_CALL( hipEventDestroy( e2 ) );
	CUDA_SAFE_CALL( hipHostFree( h_data ) );
	CUDA_SAFE_CALL( hipFree( d_data ) );

	hipDeviceReset();
}

