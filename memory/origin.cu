/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation and 
 * any modifications thereto.  Any use, reproduction, disclosure, or distribution 
 * of this software and related documentation without an express license 
 * agreement from NVIDIA Corporation is strictly prohibited.
 * 
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <cutil_inline.h>

#include "add_kernel.cu"

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char** argv)
{
	unsigned int mem_size= (1 << 25);
	float elapsed_time_in_Ms= 0;
	float bandwidth_in_MBs= 0;
	int i, max_iter= 10;
	float *h_data, *d_data;

	if( argc > 1 )
		mem_size =  (1 << atoi(argv[1]));
	unsigned int nelem= mem_size/sizeof(float);

	hipSetDevice( DEVICE );
	// allocate host memory for matrices A and B
	h_data= (float*)malloc( mem_size );
	for( i= 0; i < nelem; i++) h_data[i]= 1e0f;
	// allocate device memory
	CUDA_SAFE_CALL( hipMalloc((void**)&d_data, mem_size) );
	hipEvent_t e1, e2;
	hipEventCreate( &e1 );
	hipEventCreate( &e2 );
	// setup execution parameters
	dim3 threads( BLOCK_SIZE, 1 );
	dim3 grid( 128, 1);
	// number of elements per thread
	unsigned int nblock = nelem/(BLOCK_SIZE*grid.x);

	CUDA_SAFE_CALL( hipEventRecord( e1, 0 ) );
	for( i= 0; i < max_iter; i++ ){
		CUDA_SAFE_CALL( hipMemcpy( d_data, h_data, mem_size,
				      hipMemcpyHostToDevice) );
		add_one<<< grid, threads >>>( d_data, nblock );
		CUDA_SAFE_CALL( hipMemcpy( h_data, d_data, mem_size,
				      hipMemcpyDeviceToHost) );
	}
	CUDA_SAFE_CALL( hipEventRecord( e2, 0 ) );
	CUDA_SAFE_CALL( hipEventSynchronize( e2 ) );
	CUDA_SAFE_CALL( hipEventElapsedTime( &elapsed_time_in_Ms, e1, e2 ) );
	bandwidth_in_MBs= 1e3f * max_iter * (mem_size * 2.0f) / 
	       	(elapsed_time_in_Ms * (float)(1 << 20));
	fprintf( stdout, "origin size(MB)= %9u time(s)= %.3f bandwidth(MB/s)= %.1f\n",
		mem_size/(1<<20), elapsed_time_in_Ms/(1e3f*max_iter),
	       	bandwidth_in_MBs );

	if( check( h_data, 11e0f, nelem) == 0 )
		fprintf( stdout, "test FAILED\n" );

	// clean up memory
	CUDA_SAFE_CALL( hipEventDestroy( e1 ) );
	CUDA_SAFE_CALL( hipEventDestroy( e2 ) );
	free( h_data );
	CUDA_SAFE_CALL( hipFree( d_data ) );

	hipDeviceReset();
}

