// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include "add_kernel.cu"
#include "cuda_safe.h"

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char** argv)
{
	unsigned int mem_size= (1 << 25);
	float elapsed_time_in_Ms= 0;
	float bandwidth_in_MBs= 0;
	unsigned int i, j, max_iter= 10;
	float *h_data, *d_data;
#define NSTREAM		4
	hipStream_t stream[NSTREAM];

	if( argc > 1 )
		mem_size =  (1 << atoi(argv[1]));

	unsigned int nelem= mem_size/sizeof(float);
	int deviceCount;
	hipGetDeviceCount(&deviceCount);

	for( int d= 0; d < deviceCount; d++ ) {
	hipSetDevice( d );
	/* CUDA flags:
	hipHostMallocDefault, hipHostMallocPortable, hipHostMallocMapped,
	hipHostMallocWriteCombined */
	unsigned int flags= hipHostMallocDefault;
	// allocate host memory for matrices A and B
	CUDA_SAFE_CALL( hipHostAlloc( (void**)&h_data, mem_size, flags ) );
	for( i= 0; i < nelem; i++) h_data[i]= 1e0f;
	// allocate device memory
	CUDA_SAFE_CALL( hipMalloc((void**)&d_data, mem_size) );
	hipEvent_t e1, e2;
	hipEventCreate( &e1 );
	hipEventCreate( &e2 );
	for( j= 0; j < NSTREAM; j++ )
		hipStreamCreate( &stream[j] );
	unsigned int n_per_stream = nelem / NSTREAM;

	CUDA_SAFE_CALL( hipEventRecord( e1, 0 ) );
	for( i= 0; i < max_iter; i++ ){
		for( j= 0; j < NSTREAM; j++ ){
		CUDA_SAFE_CALL( hipMemcpyAsync( d_data+j*n_per_stream,
			h_data+j*n_per_stream,
			n_per_stream*sizeof(float),
			hipMemcpyHostToDevice, stream[j]) );
		}
		hipDeviceSynchronize();
		CUDA_SAFE_CALL( hipMemcpy( h_data, d_data, mem_size,
				      hipMemcpyDeviceToHost) );
	}
	CUDA_SAFE_CALL( hipEventRecord( e2, 0 ) );
	CUDA_SAFE_CALL( hipEventSynchronize( e2 ) );
	CUDA_SAFE_CALL( hipEventElapsedTime( &elapsed_time_in_Ms, e1, e2 ) );
	bandwidth_in_MBs= 1e3f * max_iter * (mem_size * 2.0f) / 
	       	(elapsed_time_in_Ms * (float)(1 << 20));
	fprintf( stdout, "pinned_async1 gpu= %d size(MB)= %9u time(ms)= %.3f bandwidth(MB/s)= %.1f\n",
		d, mem_size/(1<<20), elapsed_time_in_Ms/(max_iter),
	       	bandwidth_in_MBs );

	if( check( h_data, 1e0f, nelem) == 0 )
		fprintf( stdout, "test FAILED\n" );

	// clean up memory
	CUDA_SAFE_CALL( hipEventDestroy( e1 ) );
	CUDA_SAFE_CALL( hipEventDestroy( e2 ) );
	CUDA_SAFE_CALL( hipHostFree( h_data ) );
	CUDA_SAFE_CALL( hipFree( d_data ) );
	}

	hipDeviceReset();
}

