/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation and 
 * any modifications thereto.  Any use, reproduction, disclosure, or distribution 
 * of this software and related documentation without an express license 
 * agreement from NVIDIA Corporation is strictly prohibited.
 * 
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <cutil_inline.h>

#include "add_kernel.cu"

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char** argv)
{
	unsigned int mem_size= (1 << 26);
	float elapsed_time_in_Ms= 0;
	float bandwidth_in_MBs= 0;
	int i, max_iter= 10;
	float *h_data, *d_data;

	if( argc > 1 )
		mem_size =  (1 << atoi(argv[1]));

	unsigned int nelem= mem_size/sizeof(float);
	hipSetDevice( 1 );
	/* CUDA flags:
	hipHostMallocDefault, hipHostMallocPortable, hipHostMallocMapped,
	hipHostMallocWriteCombined */
	unsigned int flags= hipHostMallocWriteCombined;
	// allocate host memory for matrices A and B
	CUDA_SAFE_CALL( hipHostAlloc( (void**)&h_data, mem_size, flags ) );
	for( i= 0; i < nelem; i++) h_data[i]= 1e0f;
	// allocate device memory
	CUDA_SAFE_CALL( hipMalloc((void**)&d_data, mem_size) );
	hipEvent_t e1, e2;
	hipEventCreate( &e1 );
	hipEventCreate( &e2 );
	// setup execution parameters
	dim3 threads( BLOCK_SIZE, 1 );
	dim3 grid( nelem / threads.x, 1 );

	CUDA_SAFE_CALL( hipEventRecord( e1, 0 ) );
	for( i= 0; i < max_iter; i++ ){
		CUDA_SAFE_CALL( hipMemcpy( d_data, h_data, mem_size,
				      hipMemcpyHostToDevice) );
		add_one<<< grid, threads >>>( d_data );
		CUDA_SAFE_CALL( hipMemcpy( h_data, d_data, mem_size,
				      hipMemcpyDeviceToHost) );
	}
	CUDA_SAFE_CALL( hipEventRecord( e2, 0 ) );
	CUDA_SAFE_CALL( hipEventSynchronize( e2 ) );
	CUDA_SAFE_CALL( hipEventElapsedTime( &elapsed_time_in_Ms, e1, e2 ) );
	bandwidth_in_MBs= 1e3f * max_iter * (mem_size * 2.0f) / 
	       	(elapsed_time_in_Ms * (float)(1 << 20));
	fprintf( stdout, "pinned_wc size(MB)= %9u time(s)= %.3f bandwidth(MB/s)= %.1f\n",
		mem_size/(1<<20), elapsed_time_in_Ms/(1e3f*max_iter),
	       	bandwidth_in_MBs );

	if( check( h_data, 11e0f, nelem) == 1 )
		fprintf( stdout, "test OK\n" );
	else
		fprintf( stdout, "test FAILED\n" );

	// clean up memory
	CUDA_SAFE_CALL( hipEventDestroy( e1 ) );
	CUDA_SAFE_CALL( hipEventDestroy( e2 ) );
	CUDA_SAFE_CALL( hipHostFree( h_data ) );
	CUDA_SAFE_CALL( hipFree( d_data ) );

	hipDeviceReset();
}

