/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation and 
 * any modifications thereto.  Any use, reproduction, disclosure, or distribution 
 * of this software and related documentation without an express license 
 * agreement from NVIDIA Corporation is strictly prohibited.
 * 
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <cutil_inline.h>

#include "add_kernel.cu"

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char** argv)
{
	unsigned int mem_size= (1 << 26);
	float elapsed_time_in_Ms= 0;
	float bandwidth_in_MBs= 0;
	int i, max_iter= 10;
	char *h_data, *d_data;

	if( argc > 1 )
		mem_size =  (1 << atoi(argv[1]));

	hipSetDevice( 1 );
	/* CUDA flags:
	hipHostMallocDefault, hipHostMallocPortable, hipHostMallocMapped,
	hipHostMallocWriteCombined */
	unsigned int flags= hipHostMallocWriteCombined;
	// allocate host memory for matrices A and B
	CUDA_SAFE_CALL( hipHostAlloc( (void**)&h_data, mem_size, flags ) );
	memset( h_data, 1, mem_size );
	// allocate device memory
	CUDA_SAFE_CALL( hipMalloc((void**)&d_data, mem_size) );
	hipEvent_t e1, e2;
	hipEventCreate( &e1 );
	hipEventCreate( &e2 );
	// setup execution parameters
	dim3 threads( BLOCK_SIZE, 1 );
	dim3 grid( mem_size / threads.x, 1 );

	CUDA_SAFE_CALL( hipEventRecord( e1, 0 ) );
	for( i= 0; i < max_iter; i++ ){
		CUDA_SAFE_CALL( hipMemcpy( d_data, h_data, mem_size,
				      hipMemcpyHostToDevice) );
		add_one<<< grid, threads >>>( d_data );
		CUDA_SAFE_CALL( hipMemcpy( h_data, d_data, mem_size,
				      hipMemcpyDeviceToHost) );
	}
	CUDA_SAFE_CALL( hipEventRecord( e2, 0 ) );
	CUDA_SAFE_CALL( hipEventSynchronize( e2 ) );
	CUDA_SAFE_CALL( hipEventElapsedTime( &elapsed_time_in_Ms, e1, e2 ) );
	bandwidth_in_MBs= 1e3f * max_iter * (mem_size * 2.0f) / 
	       	(elapsed_time_in_Ms * (float)(1 << 20));
	fprintf( stdout, "pinned_wc size(MB)= %9u time(s)= %.3f bandwidth(MB/s)= %.1f\n",
		mem_size/(1<<20), elapsed_time_in_Ms/(1e3f*max_iter),
	       	bandwidth_in_MBs );

	// clean up memory
	CUDA_SAFE_CALL( hipEventDestroy( e1 ) );
	CUDA_SAFE_CALL( hipEventDestroy( e2 ) );
	CUDA_SAFE_CALL( hipHostFree( h_data ) );
	CUDA_SAFE_CALL( hipFree( d_data ) );

	hipDeviceReset();
}

