/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation and 
 * any modifications thereto.  Any use, reproduction, disclosure, or distribution 
 * of this software and related documentation without an express license 
 * agreement from NVIDIA Corporation is strictly prohibited.
 * 
 */
 
 // includes, system

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// clamp x to range [a, b]
__device__ float clamp(float x, float a, float b)
{
    return max(a, min(b, x));
}

__device__ int clamp(int x, int a, int b)
{
    return max(a, min(b, x));
}

// convert floating point rgb color to 8-bit integer
__device__ int rgbToInt(float r, float g, float b)
{
    r = clamp(r, 0.0f, 255.0f);
    g = clamp(g, 0.0f, 255.0f);
    b = clamp(b, 0.0f, 255.0f);
    return (int(b)<<16) | (int(g)<<8) | int(r);
}

// get pixel from 2D image, with clamping to border
__device__ int getPixel(unsigned int *data, int x, int y, int width, int height)
{
    x = clamp(x, 0, width-1);
    y = clamp(y, 0, height-1);
    return data[y*width+x];
}

// macros to make indexing shared memory easier
#define SMEM(X, Y) sdata[(Y)*tilew+(X)]

/*
    2D convolution using shared memory
    - operates on 8-bit RGB data stored in 32-bit int
    - assumes kernel radius is less than or equal to block size
    - not optimized for performance
     _____________
    |   :     :   |
    |_ _:_____:_ _|
    |   |     |   |
    |   |     |   |
    |_ _|_____|_ _|
  r |   :     :   |
    |___:_____:___|
      r    bw   r
    <----tilew---->
*/

__global__ void
cudaProcess(unsigned int* g_data, unsigned int* g_odata, int imgw, int imgh, int tilew, int r, float threshold, float highlight)
{
    extern __shared__ int sdata[];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bw = blockDim.x;
    int bh = blockDim.y;
    int x = blockIdx.x*bw + tx;
    int y = blockIdx.y*bh + ty;

    // copy tile to shared memory
    // center region
    SMEM(r + tx, r + ty) = getPixel(g_data, x, y, imgw, imgh);

    // borders
    if (threadIdx.x < r) {
        // left
        SMEM(tx, r + ty) = getPixel(g_data, x - r, y, imgw, imgh);
        // right
        SMEM(r + bw + tx, r + ty) = getPixel(g_data, x + bw, y, imgw, imgh);
    }
    if (threadIdx.y < r) {
        // top
        SMEM(r + tx, ty) = getPixel(g_data, x, y - r, imgw, imgh);
        // bottom
        SMEM(r + tx, r + bh + ty) = getPixel(g_data, x, y + bh, imgw, imgh);
    }

    // load corners
    if ((threadIdx.x < r) && (threadIdx.y < r)) {
        // tl
        SMEM(tx, ty) = getPixel(g_data, x - r, y - r, imgw, imgh);
        // bl
        SMEM(tx, r + bh + ty) = getPixel(g_data, x - r, y + bh, imgw, imgh);
        // tr
        SMEM(r + bw + tx, ty) = getPixel(g_data, x + bh, y - r, imgw, imgh);
        // br
        SMEM(r + bw + tx, r + bh + ty) = getPixel(g_data, x + bw, y + bh, imgw, imgh);
    }

    // wait for loads to complete
    __syncthreads();

    // perform convolution
    float rsum = 0.0;
    float gsum = 0.0;
    float bsum = 0.0;
    float samples = 0.0;

    for(int dy=-r; dy<=r; dy++) {
        for(int dx=-r; dx<=r; dx++) {
#if 0
            // try this to see the benefit of using shared memory
            int pixel = getPixel(g_data, x+dx, y+dy, imgw, imgh);
#else
            int pixel = SMEM(r+tx+dx, r+ty+dy);
#endif

            // only sum pixels within disc-shaped kernel
            float l = dx*dx + dy*dy;
            if (l <= r*r) {
                float r = float(pixel&0xff);
                float g = float((pixel>>8)&0xff);
                float b = float((pixel>>16)&0xff);
#if 1
                // brighten highlights
                float lum = (r + g + b) / (255*3);
                if (lum > threshold) {
                    r *= highlight;
                    g *= highlight;
                    b *= highlight;
                }
#endif
                rsum += r;
                gsum += g;
                bsum += b;
                samples += 1.0;
            }
        }
    }

    rsum /= samples;
    gsum /= samples;
    bsum /= samples;
    g_odata[y*imgw+x] = rgbToInt(rsum, gsum, bsum);
}

extern "C" void
launch_cudaProcess(dim3 grid, dim3 block, int sbytes, unsigned int* g_data, unsigned int* g_odata, 
					int imgw, int imgh, int tilew, 
					int radius, float threshold, float highlight)
{
    cudaProcess<<< grid, block, sbytes >>> (g_data, g_odata, imgw, imgh, block.x+(2*radius), radius, 0.8f, 4.0f);
}
