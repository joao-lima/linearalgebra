#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation and 
 * any modifications thereto.  Any use, reproduction, disclosure, or distribution 
 * of this software and related documentation without an express license 
 * agreement from NVIDIA Corporation is strictly prohibited.
 * 
 */

/*
 * This sample evaluates fair call and put prices for a
 * given set of European options by Black-Scholes formula.
 * See supplied whitepaper for more explanations.
 */



#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <string.h>
#include <cutil_inline.h>



////////////////////////////////////////////////////////////////////////////////
// Process an array of optN options on CPU
////////////////////////////////////////////////////////////////////////////////
extern "C" void BlackScholesCPU(
    float *h_CallResult,
    float *h_PutResult,
    float *h_StockPrice,
    float *h_OptionStrike,
    float *h_OptionYears,
    float Riskfree,
    float Volatility,
    int optN
);



////////////////////////////////////////////////////////////////////////////////
// Process an array of OptN options on GPU
////////////////////////////////////////////////////////////////////////////////
#include "BlackScholes_kernel.cuh"



////////////////////////////////////////////////////////////////////////////////
// Helper function, returning uniformly distributed
// random float in [low, high] range
////////////////////////////////////////////////////////////////////////////////
float RandFloat(float low, float high){
    float t = (float)rand() / (float)RAND_MAX;
    return (1.0f - t) * low + t * high;
}



////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////
const int OPT_N = 4000000;

#ifdef __DEVICE_EMULATION__
const int  NUM_ITERATIONS = 1;
#else
const int  NUM_ITERATIONS = 512;
#endif


const int          OPT_SZ = OPT_N * sizeof(float);
const float      RISKFREE = 0.02f;
const float    VOLATILITY = 0.30f;


////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv){
    //'h_' prefix - CPU (host) memory space
    float
        //Results calculated by CPU for reference
        *h_CallResultCPU,
        *h_PutResultCPU,
        //CPU copy of GPU results
        *h_CallResultGPU,
        *h_PutResultGPU,
        //CPU instance of input data
        *h_StockPrice,
        *h_OptionStrike,
        *h_OptionYears;

    //'d_' prefix - GPU (device) memory space
    float
        //Results calculated by GPU
        *d_CallResult,
        *d_PutResult,
        //GPU instance of input data
        *d_StockPrice,
        *d_OptionStrike,
        *d_OptionYears;

    double
        delta, ref, sum_delta, sum_ref, max_delta, L1norm, gpuTime;

    unsigned int hTimer;
    int i;


    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
		cutilDeviceInit(argc, argv);
	else
		hipSetDevice( cutGetMaxGflopsDeviceId() );
		
    cutilCheckError( cutCreateTimer(&hTimer) );

    printf("Initializing data...\n");
        printf("...allocating CPU memory for options.\n");
        h_CallResultCPU = (float *)malloc(OPT_SZ);
        h_PutResultCPU  = (float *)malloc(OPT_SZ);
        h_CallResultGPU = (float *)malloc(OPT_SZ);
        h_PutResultGPU  = (float *)malloc(OPT_SZ);
        h_StockPrice    = (float *)malloc(OPT_SZ);
        h_OptionStrike  = (float *)malloc(OPT_SZ);
        h_OptionYears   = (float *)malloc(OPT_SZ);

        printf("...allocating GPU memory for options.\n");
        cutilSafeCall( hipMalloc((void **)&d_CallResult,   OPT_SZ) );
        cutilSafeCall( hipMalloc((void **)&d_PutResult,    OPT_SZ) );
        cutilSafeCall( hipMalloc((void **)&d_StockPrice,   OPT_SZ) );
        cutilSafeCall( hipMalloc((void **)&d_OptionStrike, OPT_SZ) );
        cutilSafeCall( hipMalloc((void **)&d_OptionYears,  OPT_SZ) );

        printf("...generating input data in CPU mem.\n");
        srand(5347);
        //Generate options set
        for(i = 0; i < OPT_N; i++){
            h_CallResultCPU[i] = 0.0f;
            h_PutResultCPU[i]  = -1.0f;
            h_StockPrice[i]    = RandFloat(5.0f, 30.0f);
            h_OptionStrike[i]  = RandFloat(1.0f, 100.0f);
            h_OptionYears[i]   = RandFloat(0.25f, 10.0f);
        }

        printf("...copying input data to GPU mem.\n");
        //Copy options data to GPU memory for further processing
        cutilSafeCall( hipMemcpy(d_StockPrice,  h_StockPrice,   OPT_SZ, hipMemcpyHostToDevice) );
        cutilSafeCall( hipMemcpy(d_OptionStrike, h_OptionStrike,  OPT_SZ, hipMemcpyHostToDevice) );
        cutilSafeCall( hipMemcpy(d_OptionYears,  h_OptionYears,   OPT_SZ, hipMemcpyHostToDevice) );
    printf("Data init done.\n");


    printf("Executing Black-Scholes GPU kernel (%i iterations)...\n", NUM_ITERATIONS);
        cutilSafeCall( hipDeviceSynchronize() );
        cutilCheckError( cutResetTimer(hTimer) );
        cutilCheckError( cutStartTimer(hTimer) );
        for(i = 0; i < NUM_ITERATIONS; i++){
            BlackScholesGPU<<<480, 128>>>(
                d_CallResult,
                d_PutResult,
                d_StockPrice,
                d_OptionStrike,
                d_OptionYears,
                RISKFREE,
                VOLATILITY,
                OPT_N
            );
            cutilCheckMsg("BlackScholesGPU() execution failed\n");
        }
        cutilSafeCall( hipDeviceSynchronize() );
        cutilCheckError( cutStopTimer(hTimer) );
        gpuTime = cutGetTimerValue(hTimer) / NUM_ITERATIONS;
    //Both call and put is calculated
    printf("Options count             : %i     \n", 2 * OPT_N);
    printf("BlackScholesGPU() time    : %f msec\n", gpuTime);
    printf("Effective memory bandwidth: %f GB/s\n", ((double)(5 * OPT_N * sizeof(float)) * 1E-9) / (gpuTime * 1E-3));
    printf("Gigaoptions per second    : %f     \n", ((double)(2 * OPT_N) * 1E-9) / (gpuTime * 1E-3));


    printf("Reading back GPU results...\n");
        //Read back GPU results to compare them to CPU results
        cutilSafeCall( hipMemcpy(h_CallResultGPU, d_CallResult, OPT_SZ, hipMemcpyDeviceToHost) );
        cutilSafeCall( hipMemcpy(h_PutResultGPU,  d_PutResult,  OPT_SZ, hipMemcpyDeviceToHost) );


    printf("Checking the results...\n");
        printf("...running CPU calculations.\n");
        //Calculate options values on CPU
        BlackScholesCPU(
            h_CallResultCPU,
            h_PutResultCPU,
            h_StockPrice,
            h_OptionStrike,
            h_OptionYears,
            RISKFREE,
            VOLATILITY,
            OPT_N
        );

        printf("Comparing the results...\n");
        //Calculate max absolute difference and L1 distance
        //between CPU and GPU results
        sum_delta = 0;
        sum_ref   = 0;
        max_delta = 0;
        for(i = 0; i < OPT_N; i++){
            ref   = h_CallResultCPU[i];
            delta = fabs(h_CallResultCPU[i] - h_CallResultGPU[i]);
            if(delta > max_delta) max_delta = delta;
            sum_delta += delta;
            sum_ref   += fabs(ref);
        }
        L1norm = sum_delta / sum_ref;
        printf("L1 norm: %E\n", L1norm);
        printf("Max absolute error: %E\n", max_delta);
    printf((L1norm < 1e-6) ? "TEST PASSED\n" : "TEST FAILED\n");


    printf("Shutting down...\n");
        printf("...releasing GPU memory.\n");
        cutilSafeCall( hipFree(d_OptionYears)  );
        cutilSafeCall( hipFree(d_OptionStrike) );
        cutilSafeCall( hipFree(d_StockPrice)  );
        cutilSafeCall( hipFree(d_PutResult)    );
        cutilSafeCall( hipFree(d_CallResult)   );

        printf("...releasing CPU memory.\n");
        free(h_OptionYears);
        free(h_OptionStrike);
        free(h_StockPrice);
        free(h_PutResultGPU);
        free(h_CallResultGPU);
        free(h_PutResultCPU);
        free(h_CallResultCPU);
        cutilCheckError( cutDeleteTimer(hTimer) );
    printf("Shutdown done.\n");

    hipDeviceReset();

    cutilExit(argc, argv);
}
