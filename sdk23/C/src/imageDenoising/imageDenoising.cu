#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation and 
 * any modifications thereto.  Any use, reproduction, disclosure, or distribution 
 * of this software and related documentation without an express license 
 * agreement from NVIDIA Corporation is strictly prohibited.
 * 
 */



/*
 * This sample demonstrates two adaptive image denoising technqiues: 
 * KNN and NLM, based on computation of both geometric and color distance 
 * between texels. While both techniques are already implemented in the 
 * DirectX SDK using shaders, massively speeded up variation 
 * of the latter techique, taking advantage of shared memory, is implemented
 * in addition to DirectX counterparts.
 * See supplied whitepaper for more explanations.
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "imageDenoising.h"


////////////////////////////////////////////////////////////////////////////////
// Helper functions
////////////////////////////////////////////////////////////////////////////////
float Max(float x, float y){
    return (x > y) ? x : y;
}

float Min(float x, float y){
    return (x < y) ? x : y;
}

int iDivUp(int a, int b){
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

__device__ float lerpf(float a, float b, float c){
    return a + (b - a) * c;
}

__device__ float vecLen(float4 a, float4 b){
    return (
        (b.x - a.x) * (b.x - a.x) +
        (b.y - a.y) * (b.y - a.y) +
        (b.z - a.z) * (b.z - a.z)
    );
}

__device__ TColor make_color(float r, float g, float b, float a){
    return
        ((int)(a * 255.0f) << 24) |
        ((int)(b * 255.0f) << 16) |
        ((int)(g * 255.0f) <<  8) |
        ((int)(r * 255.0f) <<  0);
}



////////////////////////////////////////////////////////////////////////////////
// Global data handlers and parameters
////////////////////////////////////////////////////////////////////////////////
//Texture reference and channel descriptor for image texture
texture<uchar4, 2, hipReadModeNormalizedFloat> texImage;
hipChannelFormatDesc uchar4tex = hipCreateChannelDesc<uchar4>();

//CUDA array descriptor
hipArray *a_Src;

////////////////////////////////////////////////////////////////////////////////
// Filtering kernels
////////////////////////////////////////////////////////////////////////////////
#include "imageDenoising_copy_kernel.cu"
#include "imageDenoising_knn_kernel.cu"
#include "imageDenoising_nlm_kernel.cu"
#include "imageDenoising_nlm2_kernel.cu"

extern "C"
hipError_t CUDA_Bind2TextureArray()
{
    return hipBindTextureToArray(texImage, a_Src);
}

extern "C"
hipError_t CUDA_UnbindTexture()
{
    return hipUnbindTexture(texImage);
}

extern "C" 
hipError_t CUDA_MallocArray(uchar4 **h_Src, int imageW, int imageH)
{
    hipError_t error;

    error = hipMallocArray(&a_Src, &uchar4tex, imageW, imageH);
    error = hipMemcpyToArray(a_Src, 0, 0,
                              *h_Src, imageW * imageH * sizeof(uchar4),
                              hipMemcpyHostToDevice
                              );

    return error;
}


extern "C"
hipError_t CUDA_FreeArray()
{
    return hipFreeArray(a_Src);    
}

