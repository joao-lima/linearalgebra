#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation and 
 * any modifications thereto.  Any use, reproduction, disclosure, or distribution 
 * of this software and related documentation without an express license 
 * agreement from NVIDIA Corporation is strictly prohibited.
 * 
 *
 *
 * This sample illustrates the usage of CUDA events for both GPU timing and
 * overlapping CPU and GPU execution.  Events are insterted into a stream
 * of CUDA calls.  Since CUDA stream calls are asynchronous, the CPU can
 * perform computations while GPU is executing (including DMA memcopies
 * between the host and device).  CPU can query CUDA events to determine
 * whether GPU has completed tasks.
 *
*/

#include <stdio.h>
#include <cutil_inline.h>

__global__ void increment_kernel(int *g_data, int inc_value)
{ 
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   g_data[idx] = g_data[idx] + inc_value;
}

int correct_output(int *data, const int n, const int x)
{
    for(int i = 0; i < n; i++)
        if(data[i] != x)
            return 0;
    return 1;
}

int main(int argc, char *argv[])
{
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
	    cutilDeviceInit(argc, argv);
	else
	    hipSetDevice( cutGetMaxGflopsDeviceId());

    int n = 16 * 1024 * 1024;
    int nbytes = n * sizeof(int);
    int value = 26;

#ifdef __DEVICE_EMULATION__
    n = 1024;   // reduce workload for emulation
#endif
    
    // allocate host memory
    int *a = 0;
    cutilSafeCall( hipHostMalloc((void**)&a, nbytes) );
    memset(a, 0, nbytes);

    // allocate device memory
    int *d_a=0;
    cutilSafeCall( hipMalloc((void**)&d_a, nbytes) );
    cutilSafeCall( hipMemset(d_a, 255, nbytes) );

    // set kernel launch configuration
    dim3 threads = dim3(512, 1);
    dim3 blocks  = dim3(n / threads.x, 1);

    // create cuda event handles
    hipEvent_t start, stop;
    cutilSafeCall( hipEventCreate(&start) );
    cutilSafeCall( hipEventCreate(&stop)  );
    
    unsigned int timer;
    cutilCheckError(  cutCreateTimer(&timer)  );
    cutilCheckError(  cutResetTimer(timer)    );
    cutilSafeCall( hipDeviceSynchronize() );
    float gpu_time = 0.0f;

    // asynchronously issue work to the GPU (all to stream 0)
    cutilCheckError( cutStartTimer(timer) );
        hipEventRecord(start, 0);
        hipMemcpyAsync(d_a, a, nbytes, hipMemcpyHostToDevice, 0);
        increment_kernel<<<blocks, threads, 0, 0>>>(d_a, value);
        hipMemcpyAsync(a, d_a, nbytes, hipMemcpyDeviceToHost, 0);
        hipEventRecord(stop, 0);
    cutilCheckError( cutStopTimer(timer) );

    // have CPU do some work while waiting for stage 1 to finish
    unsigned long int counter=0;
    while( hipEventQuery(stop) == hipErrorNotReady )
    {
        counter++;
    }
    cutilSafeCall( hipEventElapsedTime(&gpu_time, start, stop) );

    // print the cpu and gpu times
    printf("time spent executing by the GPU: %.2f\n", gpu_time);
    printf("time spent by CPU in CUDA calls: %.2f\n", cutGetTimerValue(timer) );
    printf("CPU executed %d iterations while waiting for GPU to finish\n", counter);

    // check the output for correctness
    printf("--------------------------------------------------------------\n");
    if( correct_output(a, n, value) )
        printf("Test PASSED\n");
    else
        printf("Test FAILED\n");

    // release resources
    cutilSafeCall( hipEventDestroy(start) );
    cutilSafeCall( hipEventDestroy(stop) );
    cutilSafeCall( hipHostFree(a) );
    cutilSafeCall( hipFree(d_a) );

    hipDeviceReset();

    cutilExit(argc, argv);

    return 0;
}
