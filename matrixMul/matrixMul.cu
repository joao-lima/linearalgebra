#include "hip/hip_runtime.h"

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include "cuda_safe.h"

// includes, kernels
#include <matrixMul_kernel.cu>

void randomInit(float*, int);
void printDiff(float*, float*, int, int);

extern "C"
void computeGold(float*, const float*, const float*, unsigned int, unsigned int, unsigned int);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char** argv)
{
	unsigned int N= 0;
	float elapsed_time_in_Ms= 0;
	float bandwidth_in_MBs= 0;
	int i, max_iter= 10;
	float error_norm;
	float ref_norm;
	float diff;

	if( argc > 1 )
		N = atoi( argv[1] );
	else
		N = 1024;

	hipSetDevice( 0 );
	// set seed for rand()
	srand(2006);

	// allocate host memory for matrices A and B
	unsigned int size_A = WA * HA;
	unsigned int mem_size_A = sizeof(float) * size_A;
	float* h_A = (float*) malloc(mem_size_A);
	unsigned int size_B = WB * HB;
	unsigned int mem_size_B = sizeof(float) * size_B;
	float* h_B = (float*) malloc(mem_size_B);
	hipEvent_t e1, e2;

	hipEventCreate( &e1 );
	hipEventCreate( &e2 );
	// initialize host memory
	randomInit(h_A, size_A);
	randomInit(h_B, size_B);

	// allocate device memory
	float* d_A;
	CUDA_SAFE_CALL(hipMalloc((void**) &d_A, mem_size_A));
	float* d_B;
	CUDA_SAFE_CALL(hipMalloc((void**) &d_B, mem_size_B));
	// allocate device memory for result
	unsigned int size_C = WC * HC;
	unsigned int mem_size_C = sizeof(float) * size_C;
	float* d_C;
	CUDA_SAFE_CALL(hipMalloc((void**) &d_C, mem_size_C));
	// allocate host memory for the result
	float* h_C = (float*) malloc(mem_size_C);
	// setup execution parameters
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(WC / threads.x, HC / threads.y);

	CUDA_SAFE_CALL(hipEventRecord( e1, 0 ));
	for( i= 0; i < max_iter; i++ ){
		// copy host memory to device
		CUDA_SAFE_CALL(hipMemcpy(d_A, h_A, mem_size_A,
				      hipMemcpyHostToDevice) );
		CUDA_SAFE_CALL(hipMemcpy(d_B, h_B, mem_size_B,
				      hipMemcpyHostToDevice) );
		// execute the kernel
		matrixMul<<< grid, threads >>>(d_C, d_A, d_B, WA, WB);
		// check if kernel execution generated and error
		//cutilCheckMsg("Kernel execution failed");
		// copy result from device to host
		CUDA_SAFE_CALL(hipMemcpy(h_C, d_C, mem_size_C,
				      hipMemcpyDeviceToHost) );
	}
	CUDA_SAFE_CALL(hipEventRecord( e2, 0 ));
	CUDA_SAFE_CALL(hipEventSynchronize( e2 ));
	CUDA_SAFE_CALL(hipEventElapsedTime( &elapsed_time_in_Ms, e1, e2 ));
	bandwidth_in_MBs= 1e3f * max_iter * (3.0f*N*N*sizeof(float)) / 
	       	(elapsed_time_in_Ms * (float)(1 << 20));
	fprintf( stdout, "naive size= %d time(ms)= %.3f bandwidth(MB/s)= %.1f\n",
		N, elapsed_time_in_Ms/(max_iter), bandwidth_in_MBs );


	if( argc > 2 ){
		// compute reference solution
		float* h_C_ref= (float*) malloc(mem_size_C);
		computeGold(h_C_ref, h_A, h_B, HA, WA, WB);

		/* Check result against reference */
		error_norm = 0;
		ref_norm = 0;
		for (i = 0; i < N; ++i) {
			diff = h_C_ref[i] - h_C[i];
			error_norm += diff * diff;
			ref_norm += h_C_ref[i] * h_C_ref[i];
		}
		error_norm = (float)sqrt((double)error_norm);
		ref_norm = (float)sqrt((double)ref_norm);
		if (fabs(ref_norm) < 1e-7) {
			fprintf (stderr, "!!!! reference norm is 0\n");
			return EXIT_FAILURE;
		}
		printf( "Test %s\n",
			(error_norm / ref_norm < 1e-6f) ? "PASSED" : "FAILED");
		free(h_C_ref);
	}

	// clean up memory
	free(h_A);
	free(h_B);
	free(h_C);
	CUDA_SAFE_CALL( hipEventDestroy(e1) );
	CUDA_SAFE_CALL( hipEventDestroy(e2) );
	CUDA_SAFE_CALL(hipFree(d_A));
	CUDA_SAFE_CALL(hipFree(d_B));
	CUDA_SAFE_CALL(hipFree(d_C));

	hipDeviceReset();
}

// Allocates a matrix with random float entries.
void randomInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

void printDiff(float *data1, float *data2, int width, int height)
{
  int i,j,k;
  int error_count=0;
  for (j=0; j<height; j++) {
    for (i=0; i<width; i++) {
      k = j*width+i;
      if (data1[k] != data2[k]) {
         printf("diff(%d,%d) CPU=%4.4f, GPU=%4.4f n", i,j, data1[k], data2[k]);
         error_count++;
      }
    }
  }
  printf(" nTotal Errors = %d n", error_count);
}

