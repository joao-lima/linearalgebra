#include "hip/hip_runtime.h"

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include "hipblas.h"
#include "cuda_safe.h"

void randomInit(float*, int);

extern "C"
static void simple_sgemm(int n, float alpha, const float *A, const float *B,
                         float beta, float *C);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char** argv)
{
	unsigned int N= 0;
	float elapsed_time_in_Ms= 0;
	float bandwidth_in_MBs= 0;
	unsigned int i, max_iter= 10;
	float alpha = 1.0f;
	float beta = 0.0f;
	float error_norm;
	float ref_norm;
	float diff;

	if( argc > 1 )
		N = atoi( argv[1] );
	else
		N = 1024;

	hipSetDevice( 0 );
    	CUBLAS_SAFE_CALL( cublasInit() );

	// allocate host memory
	unsigned int size_A = N * N;
	unsigned int mem_size_A = sizeof(float) * size_A;
	float* h_A = (float*) malloc(mem_size_A);
	unsigned int size_B = N * N;
	unsigned int mem_size_B = sizeof(float) * size_B;
	float* h_B = (float*) malloc(mem_size_B);
	unsigned int size_C = N * N;
	unsigned int mem_size_C = sizeof(float) * size_C;
	float* h_C = (float*) malloc(mem_size_C);
	// initialize host memory
	srand(2006);
	randomInit(h_A, size_A);
	randomInit(h_B, size_B);
	randomInit(h_C, size_C);

	// allocate CUBLAS device memory 
	float *d_A, *d_B, *d_C;
	CUBLAS_SAFE_CALL( cublasAlloc( size_A, sizeof(d_A[0]), (void**)&d_A) );
	CUBLAS_SAFE_CALL( cublasAlloc( size_B, sizeof(d_B[0]), (void**)&d_B) );
	CUBLAS_SAFE_CALL( cublasAlloc( size_C, sizeof(d_C[0]), (void**)&d_C) );
	CUBLAS_SAFE_CALL( hipblasSetVector( size_A, sizeof(h_A[0]), h_A, 1, d_A, 1 ) );
	CUBLAS_SAFE_CALL( hipblasSetVector( size_B, sizeof(h_B[0]), h_B, 1, d_B, 1 ) );
	CUBLAS_SAFE_CALL( hipblasSetVector( size_C, sizeof(h_C[0]), h_C, 1, d_C, 1 ) );

	// events
	hipEvent_t e1, e2;
	hipEventCreate( &e1 );
	hipEventCreate( &e2 );

	CUDA_SAFE_CALL(hipEventRecord( e1, 0 ));
	for( i= 0; i < max_iter; i++ ){
		/* Performs operation using cublas */
		hipblasSgemm('n', 'n', N, N, N, alpha, d_A, N, d_B, N,
				beta, d_C, N);
		CUBLAS_SAFE_THREAD_SYNC();
	}
	CUDA_SAFE_CALL(hipEventRecord( e2, 0 ));
	CUDA_SAFE_CALL(hipEventSynchronize( e2 ));
	CUDA_SAFE_CALL(hipEventElapsedTime( &elapsed_time_in_Ms, e1, e2 ));
	bandwidth_in_MBs= 1e3f * max_iter * (3.0f*N*N*sizeof(float)) / 
	       	(elapsed_time_in_Ms * (float)(1 << 20));
	fprintf( stdout, "size= %d time(ms)= %.3f bandwidth(MB/s)= %.1f\n",
		N, elapsed_time_in_Ms/(max_iter), bandwidth_in_MBs );


	if( argc > 2 ){
		// compute reference solution
		float* h_C_ref= (float*) malloc(mem_size_C);
		simple_sgemm(N, alpha, h_A, h_B, beta, h_C_ref);

		/* Check result against reference */
		error_norm = 0;
		ref_norm = 0;
		for (i = 0; i < N; ++i) {
			diff = h_C_ref[i] - h_C[i];
			error_norm += diff * diff;
			ref_norm += h_C_ref[i] * h_C_ref[i];
		}
		error_norm = (float)sqrt((double)error_norm);
		ref_norm = (float)sqrt((double)ref_norm);
		if (fabs(ref_norm) < 1e-7) {
			fprintf (stderr, "!!!! reference norm is 0\n");
			return EXIT_FAILURE;
		}
		printf( "Test %s\n",
			(error_norm / ref_norm < 1e-6f) ? "PASSED" : "FAILED");

		free(h_C_ref);
	}

	// clean up memory
	free(h_A);
	free(h_B);
	free(h_C);
	CUBLAS_SAFE_CALL( cublasFree(d_A) );
	CUBLAS_SAFE_CALL( cublasFree(d_B) );
	CUBLAS_SAFE_CALL( cublasFree(d_C) );
	CUDA_SAFE_CALL( hipEventDestroy(e1) );
	CUDA_SAFE_CALL( hipEventDestroy(e2) );

	CUBLAS_SAFE_CALL( cublasShutdown() );
	hipDeviceReset();
}

// Allocates a matrix with random float entries.
void randomInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

void printDiff(float *data1, float *data2, int width, int height)
{
  int i,j,k;
  int error_count=0;
  for (j=0; j<height; j++) {
    for (i=0; i<width; i++) {
      k = j*width+i;
      if (data1[k] != data2[k]) {
         printf("diff(%d,%d) CPU=%4.4f, GPU=%4.4f n", i,j, data1[k], data2[k]);
         error_count++;
      }
    }
  }
  printf(" nTotal Errors = %d n", error_count);
}

/* Host implementation of a simple version of sgemm */
static void simple_sgemm(int n, float alpha, const float *A, const float *B,
                         float beta, float *C)
{
    int i;
    int j;
    int k;
    for (i = 0; i < n; ++i) {
        for (j = 0; j < n; ++j) {
            float prod = 0;
            for (k = 0; k < n; ++k) {
                prod += A[k * n + i] * B[j * n + k];
            }
            C[j * n + i] = alpha * prod + beta * C[j * n + i];
        }
    }
}

