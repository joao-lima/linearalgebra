
#include <iostream>
#include <algorithm>

#include "cuda_defs.h"

#include "saxpy_kernel.cu"

void check( float A, float *x, float *y, float *ref_y, unsigned int N )
{
	int result;

	std::transform( x, x+N, ref_y, ref_y, saxpy_gpu(A) );
	result= compareL2fe( ref_y, y, N, 1e-6f );
	if( result == 0 ) {
		fprintf( stdout, "ERROR\n" );
		fprintf( stdout, "%f %f\n", y[0], ref_y[0] );
		fprintf( stdout, "%f %f\n", y[44], ref_y[44] );
		fprintf( stdout, "%f %f\n", y[N-1], ref_y[N-1] );
	}
}

void saxpy( float A, float *x, float *y, unsigned int N )
{
	float *d_x, *d_y;
	int i, max_iter= 1;
	hipEvent_t e1, e2;
	dim3 threads( BLOCK_SIZE, 1 );
	dim3 grid( (N+BLOCK_SIZE-1)/BLOCK_SIZE, 1 );
	float elapsed_time_in_Ms= 0;
	float bandwidth_in_MBs= 0;
	unsigned int mem_size= N * sizeof(float);

	// Y <- A * X + Y
	// setup execution parameters
	hipEventCreate( &e1 );
	hipEventCreate( &e2 );
	CUDA_SAFE_CALL( hipMalloc((void**)&d_x, mem_size) );
	CUDA_SAFE_CALL( hipMalloc((void**)&d_y, mem_size) );

	CUDA_SAFE_CALL( hipEventRecord( e1, 0 ) );
	for( i= 0; i < max_iter; i++ ){
		CUDA_SAFE_CALL( hipMemcpy( d_x, x, N*sizeof(float),
				      hipMemcpyHostToDevice) );
		CUDA_SAFE_CALL( hipMemcpy( d_y, y, N*sizeof(float),
				      hipMemcpyHostToDevice) );
		saxpy_kernel<<< grid, threads >>>( d_x, d_y, N,
			saxpy_gpu(A) );
		CUDA_SAFE_CALL( hipMemcpy( y, d_y, N*sizeof(float),
				      hipMemcpyDeviceToHost) );
	}
	CUDA_SAFE_CALL( hipEventRecord( e2, 0 ) );
	CUDA_SAFE_CALL( hipEventSynchronize( e2 ) );

	CUDA_SAFE_CALL( hipEventElapsedTime( &elapsed_time_in_Ms, e1, e2 ) );
	bandwidth_in_MBs= 1e3f * max_iter * (mem_size * 3.0f) / 
	       	(elapsed_time_in_Ms * (float)(1 << 20));
	fprintf( stdout, "saxpy n=%d size(MB)= %9u time(ms)= %.3f bandwidth(MB/s)= %.1f\n",
		N, mem_size/(1<<20), elapsed_time_in_Ms/max_iter,
	       	bandwidth_in_MBs );
	

	// clean up memory
	CUDA_SAFE_CALL( hipEventDestroy( e1 ) );
	CUDA_SAFE_CALL( hipEventDestroy( e2 ) );
	CUDA_SAFE_CALL( hipFree(d_x) );
	CUDA_SAFE_CALL( hipFree(d_y) );
}

void randomInit( float* data, int size )
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

int main( int argc, char *argv[] )
{
	unsigned int mem_size= (1 << 25);
	float *x, *y, *ref_y;

	if( argc > 1 )
		mem_size =  (1 << atoi(argv[1]));
	unsigned int nelem= mem_size/sizeof(float);

	x= (float*) malloc( mem_size );
	y= (float*) malloc( mem_size );
	ref_y= (float*) malloc( mem_size );
	randomInit( x, nelem );
	randomInit( y, nelem );
	memcpy( ref_y, y, mem_size );
	hipSetDevice( DEVICE );
	saxpy( 2.0, x, y, nelem );
	check( 2.0, x, y, ref_y, nelem );
	free( x );
	free( y );
	free( ref_y );
	hipDeviceReset();
}

